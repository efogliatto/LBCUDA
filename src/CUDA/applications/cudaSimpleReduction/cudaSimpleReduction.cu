/*

  simpleReduction

  Suma de componentes de funciones de distribuci\'on por nodo

 */



#include <latticeMesh.h>

#include <basic.h>

#include <stdio.h>

#include <hip/hip_runtime.h>



int main(int argc, char** argv) {

    
    printf( "                    \n" );
    printf( "     o-----o-----o  \n" );
    printf( "     | -   |   - |  \n" );
    printf( "     |   - | -   |                simpleReduction\n" );
    printf( "     o<----o---->o  \n" );
    printf( "     |   - | -   |  Operacion de reduccion sobre funcion de distribucion\n" );
    printf( "     | -   |   - |  \n" );
    printf( "     o-----o-----o  \n\n" );


    // Argumentos:
    //
    // - argv[1] = numero de iteraciones
    // - argv[2] = tamanio de bloque

    uint nit = atoi( argv[1] );

    uint blsize = atoi( argv[2] );
    


    // Lectura de malla

    basicMesh mesh = readBasicMesh();


    /* // Alocacion de funcion de distribucion como arreglo unidimensional */
    /* // */
    /* // Si se desea acceder a los componentes de field usando dos indices, */
    /* // entonces puede hacerse algo como */
    /* // */
    /* // for( i = 0 ; i < mesh.nPoints ; i++) */
    /* //     for( j = 0 ; j < mesh.Q ; j++) */
    /* //         idx = i*mesh.Q + j; */

    /* uint fsize = mesh.nPoints * mesh.Q; */
    
    /* cuscalar* field = (cuscalar*)malloc( fsize * sizeof(cuscalar) ); */


    /* // Alocacion de arreglo de salida */

    /* cuscalar* sum = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); */


    
    /* // Inicializacion (puede ser otra) */

    /* for( uint i = 0 ; i < fsize ; i++ ) */
    /* 	field[i] = i; */





    
    /* // Alocacion de memoria en el device y copia */

    /* cuscalar* deviceField; */

    /* hipMalloc( (void**)&deviceField, fsize*sizeof(cuscalar) ); */


    /* cuscalar* deviceSum; */

    /* hipMalloc( (void**)&deviceSum, mesh.nPoints*sizeof(cuscalar) ); */

    



    /* // Reduccion */

    /* printf(" Aplicando reducción en %d iteraciones\n\n", nit); */

    /* timeInfo Time; */

    /* startTime(&Time); */

    /* for( uint k = 0 ; k < nit ; k++ ) { */

	

    /* } */


    /* printf( "\n   Reduccion finalizada en %f segundos\n\n", elapsedTime(&Time) ); */




    
    /* // Limpieza de memoria */

    /* free( field ); */

    /* free( sum ); */

    /* freeBasicMesh( &mesh ); */

    /* hipFree( deviceField ); */

    /* hipFree( deviceSum );     */
    
    return 0;

}
