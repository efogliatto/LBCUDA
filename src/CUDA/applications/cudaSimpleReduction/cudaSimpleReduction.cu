#include "hip/hip_runtime.h"
/*

  simpleReduction

  Suma de componentes de funciones de distribuci\'on por nodo

 */


extern "C" {
    
#include <latticeMesh.h>

#include <basic.h>

}

#include <stdio.h>

#include <hip/hip_runtime.h>

#include <cudaTest.h>

#include <cudaLatticeMesh.h>

#include <math.h>





int main(int argc, char** argv) {


    
    
    printf( "                    \n" );
    printf( "     o-----o-----o  \n" );
    printf( "     | -   |   - |  \n" );
    printf( "     |   - | -   |                cuSimpleReduction\n" );
    printf( "     o<----o---->o  \n" );
    printf( "     |   - | -   |  Operacion de reduccion en GPU  sobre funcion de distribucion\n" );
    printf( "     | -   |   - |  \n" );
    printf( "     o-----o-----o  \n\n" );


    // Argumentos:
    //
    // - argv[1] = numero de iteraciones
    // - argv[2] = xgrid

    uint nit  = atoi( argv[1] );

    int xgrid = atoi( argv[2] );
    


    // Informacion sobre el device

    hipDeviceProp_t prop;

    {
	int count;
	
	hipGetDeviceCount( &count );
	
	for (int i=0; i< count; i++)
	    hipGetDeviceProperties( &prop, i );

	printf( "\n -- Informacion general del device --  \n\n" );
	printf( "  Nombre: %s\n", prop.name );
	printf( "  Compute capability: %d.%d\n", prop.major, prop.minor );
	printf( "  Total global mem: %.2f GB\n", (float)prop.totalGlobalMem / 1000000000 );
	printf( "  Total constant Mem: %ld\n", prop.totalConstMem );
	printf( "\n\n" );

    }
    

    

    // Lectura de malla

    basicMesh mesh = readBasicMesh();

    cudaBasicMesh cmesh;

    hostToDeviceMesh( &cmesh, &mesh );




    // Alocacion de funcion de distribucion como arreglo unidimensional
    //
    // Si se desea acceder a los componentes de field usando dos indices,
    // entonces puede hacerse algo como
    //
    // for( i = 0 ; i < mesh.nPoints ; i++)
    //     for( j = 0 ; j < mesh.Q ; j++)
    //         idx = i*mesh.Q + j;

    uint fsize = mesh.nPoints * mesh.Q;
    
    cuscalar* field = (cuscalar*)malloc( fsize * sizeof(cuscalar) );


    
    // Alocacion de arreglo de salida

    cuscalar* sum = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) );


    
    // Inicializacion (puede ser otra)

    for( uint i = 0 ; i < fsize ; i++ )
    	field[i] = i;

    
    for( uint i = 0 ; i < mesh.nPoints ; i++ )
    	sum[i] = 1.0;


    
    // Alocacion de memoria en el device y copia

    cuscalar* deviceField;

    hipMalloc( (void**)&deviceField, fsize*sizeof(cuscalar) );

    hipMemcpy( deviceField, field, fsize*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceSum;

    hipMalloc( (void**)&deviceSum, mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceSum, sum, mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );    
    



    

    // Reduccion

    printf(" Aplicando reducción en %d iteraciones\n\n", nit);

    timeInfo Time;

    startTime(&Time);

    for( uint k = 0 ; k < nit ; k++ ) {
	
    	/* zerothMoment<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>(deviceField, deviceSum, cmesh.nPoints, cmesh.Q); 
    	threadedZerothMoment<<<xgrid,ceil((mesh.nPoints+1)/xgrid)>>>(deviceField, deviceSum, cmesh.nPoints, cmesh.Q);*/
	thomasModel<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>(deviceField, deviceSum, cmesh.nPoints, cmesh.Q);

	hipDeviceSynchronize();
	
    }


    scalar elap = elapsedTime(&Time);
    
    printf( "\n   Reduccion finalizada en %f segundos\n", elap );
    printf( "\n   Tiempo promedio de ejecución por iteración: %g segundos\n\n", elap/nit );    
    


    
    // Resultados vuelta al host
    
    cuscalar* dsum = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) );

    hipMemcpy( dsum, deviceSum, mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost );


    

    // Verificacion de calculo contra version de CPU

    zerothMomentCPU(sum, field, &mesh);

    {
	
	uint eq = 0;

	for(uint i = 0 ; i < mesh.nPoints ; i++) {

	    if(dsum[i] != sum[i])
		eq = 1;

	}

	if(eq != 0)
	    printf( " Los resultados de host y device difieren!\n " );
    
    }

    
    // Limpieza de memoria

    free( field );

    free( sum );

    freeBasicMesh( &mesh );

    hipFree( deviceField );

    hipFree( deviceSum );
    
    return 0;

}
