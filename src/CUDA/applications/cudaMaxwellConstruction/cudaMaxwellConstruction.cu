#include "hip/hip_runtime.h"
/*

  Maxwell construction

  Construccion de Maxwell isotermica

 */


extern "C" {
    
#include <latticeMesh.h>

#include <basic.h>

#include <exampleModel.h>   

#include <momentoFunciondist.h>   

#include <momentoVelocity.h>

#include <fuerza.h>

#include <IO.h>

}

#include <stdio.h>

#include <hip/hip_runtime.h>

#include <cudaTest.h>

#include <cudaLatticeMesh.h>

#include <cudaExampleModel.h>

#include <cudaMomentoFunciondist.h>

#include <cudaFuerza.h>

#include <math.h>





int main(int argc, char** argv) {


    
    
    printf( "                    \n" );
    printf( "     o-----o-----o  \n" );
    printf( "     | -   |   - |  \n" );
    printf( "     |   - | -   |                cudaMaxwellConstruction \n" );
    printf( "     o<----o---->o  \n" );
    printf( "     |   - | -   |  Construccion de Maxwell isotérmica en dominio bidimensional\n" );
    printf( "     | -   |   - |  \n" );
    printf( "     o-----o-----o  \n\n" );


    // Argumentos:
    //
    // - argv[1] = Pasos de tiempo
    // - argv[2] = Intervalo de escritura
    // - argv[3] = xgrid

    
    /* int xgrid = atoi( argv[3] ); */


    /* // Pasos de tiempo */
    
    /* uint timeSteps = atoi( argv[1] ); */

    /* uint wrtInterval = atoi( argv[2] ); */

    /* uint nwrite = (uint)(timeSteps/wrtInterval + 1); */

    /* uint* timeList = (uint*)malloc( nwrite * sizeof(uint) ); */
    
    /* for(int i = 0; i < nwrite; i++) */
    /*     timeList[i] = i*wrtInterval;   */
            
    /* cuscalar delta_t = 1.0; */
    


    // Informacion sobre el device

    hipDeviceProp_t prop;

    {
	int count;
	
	hipGetDeviceCount( &count );
	
	for (int i=0; i< count; i++)
	    hipGetDeviceProperties( &prop, i );

	printf( "\n -- Informacion general del device --  \n\n" );
	printf( "  Nombre: %s\n", prop.name );
	printf( "  Compute capability: %d.%d\n", prop.major, prop.minor );
	printf( "  Total global mem: %.2f GB\n", (float)prop.totalGlobalMem / 1000000000 );
	printf( "  Total constant Mem: %ld\n", prop.totalConstMem );
	printf( "\n\n" );

    }



    /* // Inicializacion de tiempo */

    /* timeInfo Time; */

    /* startTime(&Time); */


    

    /* // Parametros del modelo */

    /* cuscalar G = -1.0; */

    /* cuscalar c = 1.0; */

    /* cuscalar sigma = 0.125; */
    

    
    /* // Constantes de EOS */

    /* cuscalar a = 0.5; */

    /* cuscalar b = 4; */

    
    
    /* // Gravedad */

    /* scalar g[3] = {0,0,0}; */

    

    /* // Lectura de malla */

    /* basicMesh mesh = readBasicMesh(); */

    /* cudaBasicMesh cmesh; */

    /* hostToDeviceMesh( &cmesh, &mesh ); */
    






    /* // Alocacion de funcion de distribucion como arreglo unidimensional */
    /* // */
    /* // Si se desea acceder a los componentes de field usando dos indices, */
    /* // entonces puede hacerse algo como */
    /* // */
    /* // for( i = 0 ; i < mesh.nPoints ; i++) */
    /* //     for( j = 0 ; j < mesh.Q ; j++) */
    /* //         idx = i*mesh.Q + j; */

    /* uint fsize = mesh.nPoints * mesh.Q; */
    
    /* cuscalar* field = (cuscalar*)malloc( fsize * sizeof(cuscalar) ); */
    
    
    /* // Alocacion de arreglo de salida */

    /* cuscalar* rho = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); //Density */

    /* cuscalar* U = (cuscalar*)malloc( 3 * mesh.nPoints * sizeof(cuscalar) ); // Velocity macroscopic */

    /* cuscalar* Temp = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); // Temperature */

    /* cuscalar* fint = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) ); // Interaction force */

    /* cuscalar* f = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) ); // Total force ( volumetric add interaction ) */

       
    /* // Inicializacion de f */

    /* for( uint i = 0 ; i < fsize ; i++ ) */
    /*     field[i] = 0.0; */



    /* // Inicializacion de densidad */

    /* for( uint i = 0 ; i < mesh.nPoints ; i++ ) { */

    /* 	/\* rho[i] = (1.0 / 12.0) + (rand() % (3)-1)*0.01*1.0/12.0; *\/ */

    /* 	if( mesh.points[i][1] < 3 ) { */

    /* 	    rho[i] = 0.07; */

    /* 	} */

    /* 	else { */

    /* 	    rho[i] = 0.09; */

    /* 	} */


    /* } */
   



    /* // Inicializacion de velocidad */
    
    /* for( uint i = 0 ; i < mesh.nPoints ; i++ ){ */
	
    /* 	for( uint j = 0 ; j < 3 ; j++ ) { */
	    
    /* 	    U[i*3+j] = 0; */

    /* 	} */

    /* } */


    /* // Inicializacion de Temperatura */

    /* for( uint i = 0 ; i < mesh.nPoints ; i++ ) */
    /* 	Temp[i] = 0.9 / 27.0; */
 



    /* // Inicializacion de fuerzas */

    /* fuerzaFuerzaint(fint, rho, Temp , &mesh, G, c, mesh.lattice.cs2, a, b); */

    /* fuerzaFuerzatotal(f, fint, rho, g, &mesh); */


    /* // Asignacion de distribucion de equilibrio */
    
    /* momentoFeq( &mesh, field, rho, U); */


    
    
    /* // Alocacion de memoria en el device y copia */

    /* cuscalar* deviceField; */

    /* hipMalloc( (void**)&deviceField, fsize*sizeof(cuscalar) ); */

    /* hipMemcpy( deviceField, field, fsize*sizeof(cuscalar), hipMemcpyHostToDevice ); */


    /* cuscalar* deviceRho; */

    /* hipMalloc( (void**)&deviceRho, mesh.nPoints*sizeof(cuscalar) ); */

    /* hipMemcpy( deviceRho, rho, mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice ); */


    /* cuscalar* deviceU; */

    /* hipMalloc( (void**)&deviceU, 3*mesh.nPoints*sizeof(cuscalar) ); */

    /* hipMemcpy( deviceU, U, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice ); */


    /* cuscalar* deviceT; */

    /* hipMalloc( (void**)&deviceT, mesh.nPoints*sizeof(cuscalar) ); */

    /* hipMemcpy( deviceT, Temp, mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice ); */


    /* cuscalar* deviceFint; */

    /* hipMalloc( (void**)&deviceFint, 3*mesh.nPoints*sizeof(cuscalar) ); */

    /* hipMemcpy( deviceFint, fint, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice ); */


    /* cuscalar* deviceF; */

    /* hipMalloc( (void**)&deviceF, 3*mesh.nPoints*sizeof(cuscalar) ); */

    /* hipMemcpy( deviceF, f, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice ); */




    /* // Factores de relajacion para colision */

    /* momentoModelCoeffs relax; */

    /* relax.Tau[0] = 1.0; */
    /* relax.Tau[1] = 0.8; */
    /* relax.Tau[2] = 1.1; */
    /* relax.Tau[3] = 1.0; */
    /* relax.Tau[4] = 1.1; */
    /* relax.Tau[5] = 1.0; */
    /* relax.Tau[6] = 1.1; */
    /* relax.Tau[7] = 0.8; */
    /* relax.Tau[8] = 0.8; */

        
    /* cuscalar* deviceTau; */

    /* hipMalloc( (void**)&deviceTau, 9*sizeof(cuscalar) ); */

    /* hipMemcpy( deviceTau, relax.Tau, 9*sizeof(cuscalar), hipMemcpyHostToDevice ); */




    /* // Antes de comenzar la simulacion, escritura de los campos iniciales */

    /* char scfields[2][100] = {"rho", "T"}; */

    /* char vfields[1][100] = {"U"}; */

    /* updateCaseFile(scfields, 2, vfields, 1, timeList, nwrite); */
    

    /* writeMeshToEnsight( &mesh ); */

    /* writeScalarToEnsight("rho", rho, &mesh, 0); */

    /* writeScalarToEnsight("T", Temp, &mesh, 0); */

    /* writeVectorToEnsight("U", U, &mesh, 0); */

    
    

    /* // Ejecucion LB */

    /* for( uint ts = 1 ; ts < (timeSteps+1) ; ts++ ) { */


    /* 	// Colision */

    /* 	cudaMomentoCollision<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceField, deviceRho, deviceU, deviceF, deviceFint, deviceT, */
    /* 								    deviceTau, cmesh.lattice.M, cmesh.lattice.invM, cmesh.nPoints, */
    /* 								    cmesh.Q, delta_t, a, b, c, mesh.lattice.cs2, G, sigma); hipDeviceSynchronize(); */


    /* 	// Streaming */

	

	
    /* 	// Actualizacion de densidad macroscopica */
	
    /* 	cudaMomentoDensity<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceField, deviceRho, cmesh.nPoints, cmesh.Q);  hipDeviceSynchronize(); */



    /* 	// Actualizacion de fuerzas */

    /* 	cudaFuerzaFuerzaint<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceFint, deviceRho, deviceT, cmesh.nPoints, */
    /* 								   cmesh.Q,  cmesh.lattice.vel, cmesh.nb, G, c, mesh.lattice.cs2, a, b);  hipDeviceSynchronize(); */

    /* 	cudaFuerzaFuerzatotal<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceF, deviceFint, deviceRho, g, cmesh.nPoints);  hipDeviceSynchronize(); */



    /* 	// Actualizacion de velocidad macroscopica */

    /* 	cudaMomentoVelocity<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>(deviceField, deviceRho, deviceU, cmesh.lattice.vel, */
    /* 								  cmesh.nPoints, cmesh.Q, delta_t );  hipDeviceSynchronize(); */
	
	


    /* 	// Escritura de campos */
	
    /* 	for( uint wt = 0 ; wt < nwrite ; wt++ ) { */

    /* 	    if( timeList[wt] == ts ) { */


    /* 		// Copia de vuelta al host */

    /* 		hipMemcpy( rho, deviceRho, mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); */

    /* 		hipMemcpy( Temp, deviceT, mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); */

    /* 		hipMemcpy( U, deviceU, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); */
		

		
    /* 	    	scalar elap = elapsedTime(&Time); */

    /* 	    	printf( " Time = %d\n", ts ); */
		
    /* 	    	printf( " Elapsed time = %.4f segundos\n\n", elap ); */
		

    /* 	    	writeScalarToEnsight("rho", rho, &mesh, wt); */

    /* 	    	writeScalarToEnsight("T", Temp, &mesh, wt); */

    /* 	    	writeVectorToEnsight("U", U, &mesh, wt); */

    /* 	    } */

    /* 	} */

	
	
    /* } */
    

   

    
    /* // Limpieza de memoria host */

    /* freeBasicMesh( &mesh ); */

    /* free( field ); */

    /* free( rho ); */

    /* free( U ); */

    /* free( Temp ); */

    /* free( f ); */

    /* free( fint ); */

    
    /* // Limpieza de memoria device */

    /* hipFree( deviceField ); */

    /* hipFree( deviceRho ); */

    /* hipFree( deviceU ); */

    /* hipFree( deviceT ); */

    /* hipFree( deviceFint ); */

    /* hipFree( deviceF ); */

    /* hipFree( deviceTau); */

    

   
    return 0;

}
