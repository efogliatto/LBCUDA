/*

  cudaNbReduction

  Suma de componentes de funciones de distribuci\'on en los vecinos, y reduccion por nodo

 */


extern "C" {
    
#include <latticeMesh.h>

#include <basic.h>

}

#include <stdio.h>

#include <hip/hip_runtime.h>

#include <cudaTest.h>

#include <cudaLatticeMesh.h>




int main(int argc, char** argv) {


    
    
    printf( "                    \n" );
    printf( "     o-----o-----o  \n" );
    printf( "     | -   |   - |  \n" );
    printf( "     |   - | -   |                cudaNbReduction\n" );
    printf( "     o<----o---->o  \n" );
    printf( "     |   - | -   |  Operacion de reduccion en GPU sobre nodos vecinos\n" );
    printf( "     | -   |   - |  \n" );
    printf( "     o-----o-----o  \n\n" );


    // Argumentos:
    //
    // - argv[1] = numero de iteraciones

    uint nit = atoi( argv[1] );
    


    // Informacion sobre el device

    hipDeviceProp_t prop;

    {
	int count;
	
	hipGetDeviceCount( &count );
	
	for (int i=0; i< count; i++)
	    hipGetDeviceProperties( &prop, i );

	printf( "\n -- Informacion general del device --  \n\n" );
	printf( "  Nombre: %s\n", prop.name );
	printf( "  Compute capability: %d.%d\n", prop.major, prop.minor );
	printf( "  Total global mem: %.2f GB\n", (float)prop.totalGlobalMem / 1000000000 );
	printf( "  Total constant Mem: %ld\n", prop.totalConstMem );
	printf( "\n\n" );

    }
    

    

    // Lectura de malla

    basicMesh mesh = readBasicMesh();

    cudaBasicMesh cmesh;

    hostToDeviceMesh( &cmesh, &mesh );




    // Alocacion de funcion de distribucion como arreglo unidimensional
    //
    // Si se desea acceder a los componentes de field usando dos indices,
    // entonces puede hacerse algo como
    //
    // for( i = 0 ; i < mesh.nPoints ; i++)
    //     for( j = 0 ; j < mesh.Q ; j++)
    //         idx = i*mesh.Q + j;

    uint fsize = mesh.nPoints * mesh.Q;
    
    cuscalar* field = (cuscalar*)malloc( fsize * sizeof(cuscalar) );


    
    // Alocacion de arreglo de salida

    cuscalar* sum = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) );




    
    // Inicializacion (puede ser otra)

    for( uint i = 0 ; i < fsize ; i++ )
    	field[i] = i;

    
    for( uint i = 0 ; i < mesh.nPoints ; i++ )
    	sum[i] = 1.0;



    
    // Alocacion de memoria en el device y copia

    cuscalar* deviceField;

    hipMalloc( (void**)&deviceField, fsize*sizeof(cuscalar) );

    hipMemcpy( deviceField, field, fsize*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceSum;

    hipMalloc( (void**)&deviceSum, mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceSum, sum, mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );    
    





    
    /**************************************/
    /*            Neighbours              */
    /**************************************/

    int* deviceNb;

    {

	// First create one-dimensional array

	int* hostNb = (int*)malloc( mesh.nPoints * mesh.Q * sizeof(int) );

	for( uint i = 0 ; i < mesh.nPoints ; i++ ) {

	    for( uint j = 0 ; j < mesh.Q ; j++ ) {

		hostNb[ i*mesh.Q + j ] = mesh.nb[i][j];

	    }

	}

	

	// Copy to device
    
	hipMalloc( (void**)&deviceNb, mesh.nPoints*mesh.Q*sizeof(int) );

	hipMemcpy( deviceNb, hostNb, mesh.nPoints*mesh.Q*sizeof(int), hipMemcpyHostToDevice );

	free(hostNb);

    }
    
    

    // Reduccion

    printf(" Aplicando reducción en %d iteraciones\n\n", nit);

    timeInfo Time;

    startTime(&Time);

    for( uint k = 0 ; k < nit ; k++ )
    	nbReduction<<<mesh.nPoints,1>>>(deviceField, deviceSum, deviceNb, cmesh.nPoints, cmesh.Q);


    scalar elap = elapsedTime(&Time);
    
    printf( "\n   Reduccion finalizada en %f segundos\n", elap );
    printf( "\n   Tiempo promedio de ejecución por iteración: %g segundos\n\n", elap/nit );    


    

    
    // Resultados vuelta al host
    
    cuscalar* dsum = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) );

    hipMemcpy( dsum, deviceSum, mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost );
    
    

    // Verificacion de calculo contra version de CPU

    nbReductionCPU(sum, field, &mesh);

    {
	
    	uint eq = 0;

    	for(uint i = 0 ; i < mesh.nPoints ; i++) {

    	    if(dsum[i] != sum[i])
    	    	eq = 1;

    	}

    	if(eq != 0)
    	    printf( " Los resultados de host y device difieren!\n " );
    
    }

    
    // Limpieza de memoria

    free( field );

    free( sum );

    freeBasicMesh( &mesh );

    hipFree( deviceField );

    hipFree( deviceSum );
    
    return 0;

}
