#include "hip/hip_runtime.h"
/*

  simpleReduction

  Suma de componentes de funciones de distribuci\'on por nodo

 */


extern "C" {
    
#include <latticeMesh.h>

#include <basic.h>

#include <exampleModel.h>   

#include <momentoFunciondist.h>   

#include <momentoVelocity.h>

#include <fuerza.h>   

}

#include <stdio.h>

#include <hip/hip_runtime.h>

#include <cudaTest.h>

#include <cudaLatticeMesh.h>

#include <cudaExampleModel.h>

#include <cudaMomentoFunciondist.h>

#include <cudaFuerza.h>

#include <math.h>





int main(int argc, char** argv) {


    
    
    printf( "                    \n" );
    printf( "     o-----o-----o  \n" );
    printf( "     | -   |   - |  \n" );
    printf( "     |   - | -   |                cuSimpleReduction\n" );
    printf( "     o<----o---->o  \n" );
    printf( "     |   - | -   |  Operacion de reduccion en GPU  sobre funcion de distribucion\n" );
    printf( "     | -   |   - |  \n" );
    printf( "     o-----o-----o  \n\n" );


    // Argumentos:
    //
    // - argv[1] = numero de iteraciones
    // - argv[2] = xgrid

    uint nit  = atoi( argv[1] );

    int xgrid = atoi( argv[2] );
    


    // Informacion sobre el device

    hipDeviceProp_t prop;

    {
	int count;
	
	hipGetDeviceCount( &count );
	
	for (int i=0; i< count; i++)
	    hipGetDeviceProperties( &prop, i );

	printf( "\n -- Informacion general del device --  \n\n" );
	printf( "  Nombre: %s\n", prop.name );
	printf( "  Compute capability: %d.%d\n", prop.major, prop.minor );
	printf( "  Total global mem: %.2f GB\n", (float)prop.totalGlobalMem / 1000000000 );
	printf( "  Total constant Mem: %ld\n", prop.totalConstMem );
	printf( "\n\n" );

    }
    
    // Parametros del modelo D2Q9

    int a = 1;

    int b = 4;

    cuscalar G = -1.0; 

    cuscalar c = 1.0; 

    cuscalar cs_2 = ( 1.0/3.0 ); 

    scalar sigma = 1.0;

    // Aceleracion del problema

    cuscalar g[3] = {0,0,0};

    

    // Lectura de malla

    basicMesh mesh = readBasicMesh();

    cudaBasicMesh cmesh;

    hostToDeviceMesh( &cmesh, &mesh );




    // Alocacion de funcion de distribucion como arreglo unidimensional
    //
    // Si se desea acceder a los componentes de field usando dos indices,
    // entonces puede hacerse algo como
    //
    // for( i = 0 ; i < mesh.nPoints ; i++)
    //     for( j = 0 ; j < mesh.Q ; j++)
    //         idx = i*mesh.Q + j;

    uint fsize = mesh.nPoints * mesh.Q;
    
    cuscalar* field = (cuscalar*)malloc( fsize * sizeof(cuscalar) );
    
    // Alocacion de arreglo de salida

    cuscalar* rho = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); //Density

    cuscalar* U = (cuscalar*)malloc( 3 * mesh.nPoints * sizeof(cuscalar) ); // Velocity macroscopic    

    cuscalar* Temp = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); // Temperature

    cuscalar* fint = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) ); // Interaction force

    cuscalar* f = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) ); // Total force ( volumetric add interaction ) 

       
    // Inicializacion (puede ser otra)

    for( uint i = 0 ; i < fsize ; i++ )
        field[i] = 1.0;
        
    for( uint i = 0 ; i < mesh.nPoints ; i++ )
        rho[i] = 1.0;
    
    
    /*for( uint i = 0 ; i < (mesh.nPoints/2) ; i++ )
        rho[i] = 1.0;
        
    for( uint i = (mesh.nPoints/2) ; i < mesh.nPoints ; i++ )
    	rho[i] = 2.0;    */

    for( uint i = 0 ; i < (3*mesh.nPoints) ; i++ )
    	U[i] = 2.0;

    for( uint i = 0 ; i < mesh.nPoints ; i++ )
    	Temp[i] = 1e-2;

    for( uint i = 0 ; i < (3*mesh.nPoints) ; i++ )
    	fint[i] = 5.0;

    for( uint i = 0 ; i < (3*mesh.nPoints) ; i++ )
    	f[i] = 8.0;

        
    // Alocacion de memoria en el device y copia

    cuscalar* deviceField;

    hipMalloc( (void**)&deviceField, fsize*sizeof(cuscalar) );

    hipMemcpy( deviceField, field, fsize*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceRho;

    hipMalloc( (void**)&deviceRho, mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceRho, rho, mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceU;

    hipMalloc( (void**)&deviceU, 3*mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceU, U, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceT;

    hipMalloc( (void**)&deviceT, mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceT, Temp, mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceFint;

    hipMalloc( (void**)&deviceFint, 3*mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceFint, fint, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceF;

    hipMalloc( (void**)&deviceF, 3*mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceF, f, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    
    // Factores de relajacion para colision

//    exampleModelCoeffs relax;

    momentoModelCoeffs relax;

    for( uint i = 0 ; i < 9 ; i++ )
	    relax.Tau[i] = 1;


    
    cuscalar* deviceTau;

    hipMalloc( (void**)&deviceTau, 9*sizeof(cuscalar) );

    hipMemcpy( deviceTau, relax.Tau, 9*sizeof(cuscalar), hipMemcpyHostToDevice );    


    cuscalar delta_t_cu = 1.0;
    

    // Reduccion

    printf(" Colisión en %d iteraciones\n\n", nit);

    timeInfo Time;

    startTime(&Time);

    for( uint k = 0 ; k < nit ; k++ ) {
	
//    	cudaExampleCollision<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceField, deviceRho, deviceU, deviceTau, cmesh.lattice.M, cmesh.lattice.invM, cmesh.nPoints, cmesh.Q );

	    cudaMomentoCollision<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceField, deviceRho, deviceU, deviceF, deviceFint, deviceT, deviceTau, cmesh.lattice.M, cmesh.lattice.invM, cmesh.nPoints, cmesh.Q, delta_t_cu, a, b, c, cs_2, G, sigma);

        hipDeviceSynchronize();
        


//        cudaFuerzaFuerzaint<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceFint, deviceRho, deviceT, cmesh.nPoints, cmesh.Q,  cmesh.lattice.vel, cmesh.nb, G, c, cs_2, a, b)   ;

    //    hipDeviceSynchronize();

    //     cudaFuerzaFuerzatotal<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceF, deviceFint, deviceRho, g, cmesh.nPoints);	

    //	hipDeviceSynchronize();
/*
        cudaMomentoVelocity<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>(deviceField, deviceRho, deviceU, cmesh.lattice.vel, cmesh.nPoints, cmesh.Q, delta_t_cu );

    	hipDeviceSynchronize();

        cudaMomentoDensity<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceField, deviceRho, cmesh.nPoints, cmesh.Q);  

    	hipDeviceSynchronize();

        */

	
    }

    scalar delta_t = 1.0;	
    scalar elap = elapsedTime(&Time);
    
    printf( "\n   Colisión finalizada en %f segundos\n", elap );
    printf( "\n   Tiempo promedio de ejecución por iteración: %g segundos\n\n", elap/nit );
    


    
    // Resultados vuelta al host
    
    cuscalar* dcol = (cuscalar*)malloc( fsize * sizeof(cuscalar) );

    hipMemcpy( dcol, deviceField, fsize*sizeof(cuscalar), hipMemcpyDeviceToHost );

/*-----------------------------------------------------------------------------------------*/
/*
    cuscalar* dRho = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) );

    hipMemcpy( dRho, deviceRho, mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 


    cuscalar* dU = (cuscalar*)malloc( 3 * mesh.nPoints * sizeof(cuscalar) );

    hipMemcpy( dU, deviceU, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 


    cuscalar* dT = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); 

    hipMemcpy( dT, deviceT, mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 


 */ cuscalar* dFint = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) );

    hipMemcpy( dFint, deviceFint, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 
/*

    cuscalar* dF = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) );

    hipMemcpy( dF, deviceF, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 

*/
   
/*-----------------------------------------------------------------------------------------*/



    // Verificacion de calculo contra version de CPU

//    exampleCollision( &mesh, &relax, field, rho, U );

    momentoCollision( &mesh, &relax, field, rho, U, f, fint, Temp, delta_t, a, b, c, cs_2, G, sigma);

							      // A continuacion se calculan el resto de los parametroz para ir actualizandolos
    

    
    //fuerzaFuerzaint(fint, rho, Temp , &mesh, G, c, cs_2, a, b);
/*
    fuerzaFuerzatotal(f, fint, rho, g, &mesh); 

    momentoVelocity( rho,  U, field, &mesh, delta_t, f);

    momentoDensity( rho, field, &mesh);  

    fuerzaS(S, f, fint, U, psi, sigma, relax.Tau, &mesh, delta_t);
*/

    {
	
    	uint eq = 0;

        printf( "    CUDA \t \t     C \t \t \t    DIFIEREN \n\n");

    	for(uint i = 0 ; i < fsize ; i++) {
        //for(uint i = 0 ; i <   mesh.nPoints*3 ; i++) {            
            eq = 0;
            cuscalar diferencia = fabs(dcol[i] - field[i]);
          
            if( diferencia > 0.000001 )
                eq = 1;


            //if(dcol[i] != field[i])
              //  eq = 1;
            printf( "%lf \t %lf \t \t \t %d \n", dcol[i],field[i],eq);
    	    //printf( "%f \t  %f  \t \t \t %d \n\n", dFint[i],fint[i],eq);
//	    printf( "%d \n", eq);	
//	    printf( "%f \n", S[i]);	

    	}

    	if(eq != 0)
    	    printf( "\n Los resultados de host y device difieren!\n\n " );
    
    }

    printf( "\n\n Comienza la limpieza de memoria!\n\n " );

    
    // Limpieza de memoria host

    freeBasicMesh( &mesh );

    free( field );

    free( rho );

    free( U ); 

    free( Temp );

    free( f );   

    free( fint );

    // Limpieza de memoria device

    hipFree( deviceField );

    hipFree( deviceRho );

    hipFree( deviceU );

    hipFree( deviceT );

    hipFree( deviceFint );

    hipFree( deviceF );

    hipFree( deviceTau);

    // Limpieza de memoria host de comparacion

    free( dcol );


    
printf( "\n\n Finaliza la limpieza de memoria!\n\n " );

   
    return 0;

}
