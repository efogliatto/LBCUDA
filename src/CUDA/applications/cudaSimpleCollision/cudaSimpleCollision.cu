#include "hip/hip_runtime.h"
/*

  simpleReduction

  Suma de componentes de funciones de distribuci\'on por nodo

 */


extern "C" {
    
#include <latticeMesh.h>

#include <basic.h>

#include <exampleModel.h>   

#include <momentoFunciondist.h>   

#include <momentoVelocity.h>

#include <fuerza.h>   

}

#include <stdio.h>

#include <hip/hip_runtime.h>

#include <cudaTest.h>

#include <cudaLatticeMesh.h>

#include <cudaExampleModel.h>

#include <cudaMomentoFunciondist.h>

#include <math.h>





int main(int argc, char** argv) {


    
    
    printf( "                    \n" );
    printf( "     o-----o-----o  \n" );
    printf( "     | -   |   - |  \n" );
    printf( "     |   - | -   |                cuSimpleReduction\n" );
    printf( "     o<----o---->o  \n" );
    printf( "     |   - | -   |  Operacion de reduccion en GPU  sobre funcion de distribucion\n" );
    printf( "     | -   |   - |  \n" );
    printf( "     o-----o-----o  \n\n" );


    // Argumentos:
    //
    // - argv[1] = numero de iteraciones
    // - argv[2] = xgrid

    uint nit  = atoi( argv[1] );

    int xgrid = atoi( argv[2] );
    


    // Informacion sobre el device

    hipDeviceProp_t prop;

    {
	int count;
	
	hipGetDeviceCount( &count );
	
	for (int i=0; i< count; i++)
	    hipGetDeviceProperties( &prop, i );

	printf( "\n -- Informacion general del device --  \n\n" );
	printf( "  Nombre: %s\n", prop.name );
	printf( "  Compute capability: %d.%d\n", prop.major, prop.minor );
	printf( "  Total global mem: %.2f GB\n", (float)prop.totalGlobalMem / 1000000000 );
	printf( "  Total constant Mem: %ld\n", prop.totalConstMem );
	printf( "\n\n" );

    }
    
    // Parametros del modelo D2Q9

    int a = 1;

    int b = 4;

    cuscalar G = -1.0; 

    cuscalar c = 1.0; 

    cuscalar cs_2 = ( 1/3 ); 

    scalar sigma = 1.0;

    // Aceleracion del problema

    cuscalar g[3] = {0,0,0};

    

    // Lectura de malla

    basicMesh mesh = readBasicMesh();

    cudaBasicMesh cmesh;

    hostToDeviceMesh( &cmesh, &mesh );




    // Alocacion de funcion de distribucion como arreglo unidimensional
    //
    // Si se desea acceder a los componentes de field usando dos indices,
    // entonces puede hacerse algo como
    //
    // for( i = 0 ; i < mesh.nPoints ; i++)
    //     for( j = 0 ; j < mesh.Q ; j++)
    //         idx = i*mesh.Q + j;

    uint fsize = mesh.nPoints * mesh.Q;
    
    cuscalar* field = (cuscalar*)malloc( fsize * sizeof(cuscalar) );


    
    // Alocacion de arreglo de salida

    cuscalar* rho = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); //Density

    cuscalar* U = (cuscalar*)malloc( 3 * mesh.nPoints * sizeof(cuscalar) ); // Velocity macroscopic    

    // Alocacion de arreglo de otros parametros

    cuscalar* T = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); // Temperature

    cuscalar* p = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); // Presion

    cuscalar* psi = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); // Arreglo con la funcion psi calculada

    cuscalar* fint = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) ); // Interaction force

    cuscalar* f = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) ); // Total force ( volumetric add interaction ) 

    cuscalar* S = (cuscalar*)malloc( mesh.nPoints * mesh.Q * sizeof(cuscalar) ); // Source term

    
    // Inicializacion (puede ser otra)

    for( uint i = 0 ; i < fsize ; i++ )
    	field[i] = 1;

    
    for( uint i = 0 ; i < mesh.nPoints ; i++ )
    	rho[i] = 1.0;

    for( uint i = 0 ; i < (3*mesh.nPoints) ; i++ )
    	U[i] = 0;

    for( uint i = 0 ; i < mesh.nPoints ; i++ )
    	T[i] = 0.0;

    for( uint i = 0 ; i < mesh.nPoints ; i++ )
    	p[i] = 0.0;

    for( uint i = 0 ; i < mesh.nPoints ; i++ )
    	psi[i] = 0.0;

    for( uint i = 0 ; i < (3*mesh.nPoints) ; i++ )
    	fint[i] = 0.0;

    for( uint i = 0 ; i < (3*mesh.nPoints) ; i++ )
    	f[i] = 0.0;

    for( uint i = 0 ; i < (mesh.Q*mesh.nPoints) ; i++ )
    	S[i] = 0.0;

    
    // Alocacion de memoria en el device y copia

    cuscalar* deviceField;

    hipMalloc( (void**)&deviceField, fsize*sizeof(cuscalar) );

    hipMemcpy( deviceField, field, fsize*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceRho;

    hipMalloc( (void**)&deviceRho, mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceRho, rho, mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceU;

    hipMalloc( (void**)&deviceU, 3*mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceU, U, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );




    // Factores de relajacion para colision

//    exampleModelCoeffs relax;

    momentoModelCoeffs relax;

    for( uint i = 0 ; i < 9 ; i++ )
	relax.Tau[i] = 1;


    
    cuscalar* deviceTau;

    hipMalloc( (void**)&deviceTau, 9*sizeof(cuscalar) );

    hipMemcpy( deviceTau, relax.Tau, 9*sizeof(cuscalar), hipMemcpyHostToDevice );    


    cuscalar delta_t_cu = 1.0;
    

    // Reduccion

    printf(" Colisión en %d iteraciones\n\n", nit);

    timeInfo Time;

    startTime(&Time);

    for( uint k = 0 ; k < nit ; k++ ) {
	
//    	cudaExampleCollision<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceField, deviceRho, deviceU, deviceTau, cmesh.lattice.M, cmesh.lattice.invM, cmesh.nPoints, cmesh.Q );

	cudaMomentoCollision<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceField, deviceRho, deviceU, deviceTau, cmesh.lattice.M, cmesh.lattice.invM, cmesh.nPoints, cmesh.Q, delta_t_cu );

    	hipDeviceSynchronize();
	
    }

    scalar delta_t = 1.0;	
    scalar elap = elapsedTime(&Time);
    
    printf( "\n   Colisión finalizada en %f segundos\n", elap );
    printf( "\n   Tiempo promedio de ejecución por iteración: %g segundos\n\n", elap/nit );
    


    
    // Resultados vuelta al host
    
    cuscalar* dcol = (cuscalar*)malloc( fsize * sizeof(cuscalar) );

    hipMemcpy( dcol, deviceField, fsize*sizeof(cuscalar), hipMemcpyDeviceToHost );


    

    // Verificacion de calculo contra version de CPU

//    exampleCollision( &mesh, &relax, field, rho, U );
    momentoCollision( &mesh, &relax, field, rho, U, delta_t, S ); // Calculo de la funcion de distribucion con valores de los parametros seteados para inicializar

							      // A continuacion se calculan el resto de los parametroz para ir actualizandolos
    
    fuerzaPresionEOS(p, rho, T, &mesh, a, b);    

    fuerzaPsi(psi, p, rho, c, cs_2, G, &mesh);

    fuerzaFuerzaint(fint, psi, &mesh, G);

    fuerzaFuerzatotal(f, fint, rho, g, &mesh);

    momentoVelocity( rho,  U, field, &mesh, delta_t, f);

    momentoDensity( rho, field, &mesh);  

    fuerzaS(S, f, fint, U, psi, sigma, relax.Tau[1], relax.Tau[2], &mesh, delta_t);


    {
	
    	uint eq = 0;

    	for(uint i = 0 ; i < fsize ; i++) {

	    printf( "%f \t %f \n", dcol[i],field[i]);	
//	    printf( "%f \n", S[i]);	

    	    if(dcol[i] != field[i])
    		eq = 1;

    	}

    	if(eq != 0)
    	    printf( " Los resultados de host y device difieren!\n " );
    
    }

    
    // Limpieza de memoria

    free( field );

    free( rho );

    free( U ); 

    free( T );

    free( p );   

    free( psi );

    free( f );   

    free( fint );

    free( S );

    freeBasicMesh( &mesh );

    hipFree( deviceField );

    hipFree( deviceRho );

    hipFree( deviceU );
    
    
    return 0;

}
