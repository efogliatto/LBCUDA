#include "hip/hip_runtime.h"
/*

  simpleReduction

  Suma de componentes de funciones de distribuci\'on por nodo

 */


extern "C" {
    
#include <latticeMesh.h>

#include <basic.h>

#include <exampleModel.h>   

#include <momentoFunciondist.h>   

#include <momentoVelocity.h>

#include <fuerza.h>   

}

#include <stdio.h>

#include <hip/hip_runtime.h>

#include <cudaTest.h>

#include <cudaLatticeMesh.h>

#include <cudaExampleModel.h>

#include <cudaMomentoFunciondist.h>

#include <cudaFuerza.h>

#include <math.h>





int main(int argc, char** argv) {


    
    
    printf( "                    \n" );
    printf( "     o-----o-----o  \n" );
    printf( "     | -   |   - |  \n" );
    printf( "     |   - | -   |                cuSimpleReduction\n" );
    printf( "     o<----o---->o  \n" );
    printf( "     |   - | -   |  Operacion de reduccion en GPU  sobre funcion de distribucion\n" );
    printf( "     | -   |   - |  \n" );
    printf( "     o-----o-----o  \n\n" );


    // Argumentos:
    //
    // - argv[1] = numero de iteraciones
    // - argv[2] = xgrid

    uint nit  = atoi( argv[1] );

    int xgrid = atoi( argv[2] );
    


    // Informacion sobre el device

    hipDeviceProp_t prop;

    {
	int count;
	
	hipGetDeviceCount( &count );
	
	for (int i=0; i< count; i++)
	    hipGetDeviceProperties( &prop, i );

	printf( "\n -- Informacion general del device --  \n\n" );
	printf( "  Nombre: %s\n", prop.name );
	printf( "  Compute capability: %d.%d\n", prop.major, prop.minor );
	printf( "  Total global mem: %.2f GB\n", (float)prop.totalGlobalMem / 1000000000 );
	printf( "  Total constant Mem: %ld\n", prop.totalConstMem );
	printf( "\n\n" );

    }
    
    // Parametros del modelo D2Q9

    int a = 1;

    int b = 4;

    cuscalar G = -1.0; 

    cuscalar c = 1.0; 

    cuscalar cs_2 = ( 1.0/3.0 ); 

    scalar sigma = 1.0;

    // Aceleracion del problema

    cuscalar g[3] = {0,0,0};

    

    // Lectura de malla

    basicMesh mesh = readBasicMesh();

    cudaBasicMesh cmesh;

    hostToDeviceMesh( &cmesh, &mesh );




    // Alocacion de funcion de distribucion como arreglo unidimensional
    //
    // Si se desea acceder a los componentes de field usando dos indices,
    // entonces puede hacerse algo como
    //
    // for( i = 0 ; i < mesh.nPoints ; i++)
    //     for( j = 0 ; j < mesh.Q ; j++)
    //         idx = i*mesh.Q + j;

    uint fsize = mesh.nPoints * mesh.Q;
    
    cuscalar* field = (cuscalar*)malloc( fsize * sizeof(cuscalar) );


    
    // Alocacion de arreglo de salida

    cuscalar* rho = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); //Density

    cuscalar* U = (cuscalar*)malloc( 3 * mesh.nPoints * sizeof(cuscalar) ); // Velocity macroscopic    

    // Alocacion de arreglo de otros parametros

    cuscalar* T = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); // Temperature

    cuscalar* p_EOS = (cuscalar*)malloc( sizeof(cuscalar) ); // Presion

    cuscalar* psi = (cuscalar*)malloc( sizeof(cuscalar) ); // Arreglo con la funcion psi calculada	

    cuscalar* fint = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) ); // Interaction force

    cuscalar* f = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) ); // Total force ( volumetric add interaction ) 

    cuscalar* S = (cuscalar*)malloc( mesh.Q * sizeof(cuscalar) ); // Source term

    
    // Inicializacion (puede ser otra)

    for( uint i = 0 ; i < fsize ; i++ )
    	field[i] = 1;

    
    for( uint i = 0 ; i < mesh.nPoints ; i++ )
    	rho[i] = 1.0;

    for( uint i = 0 ; i < (3*mesh.nPoints) ; i++ )
    	U[i] = 0;

    for( uint i = 0 ; i < mesh.nPoints ; i++ )
    	T[i] = 0.0;

    p_EOS[0] = 0.0;


    psi[0] = 0.0;					

    for( uint i = 0 ; i < (3*mesh.nPoints) ; i++ )
    	fint[i] = 0.0;

    for( uint i = 0 ; i < (3*mesh.nPoints) ; i++ )
    	f[i] = 0.0;

/*    for( uint i = 0 ; i < (mesh.Q*mesh.nPoints) ; i++ )
    	S[i] = 0.0;						*/

    
    // Alocacion de memoria en el device y copia

    cuscalar* deviceField;

    hipMalloc( (void**)&deviceField, fsize*sizeof(cuscalar) );

    hipMemcpy( deviceField, field, fsize*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceRho;

    hipMalloc( (void**)&deviceRho, mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceRho, rho, mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceU;

    hipMalloc( (void**)&deviceU, 3*mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceU, U, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceT;

    hipMalloc( (void**)&deviceT, mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceT, T, mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceP_EOS;
 
    hipMalloc( (void**)&deviceP_EOS, sizeof(cuscalar) );

    hipMemcpy( deviceP_EOS, p_EOS, sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* devicePsi;

    hipMalloc( (void**)&devicePsi, sizeof(cuscalar) );

    hipMemcpy( devicePsi, psi, sizeof(cuscalar), hipMemcpyHostToDevice );		


    cuscalar* deviceFint;

    hipMalloc( (void**)&deviceFint, 3*mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceFint, fint, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceF;

    hipMalloc( (void**)&deviceF, 3*mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceF, f, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );


    cuscalar* deviceS;

    hipMalloc( (void**)&deviceS, mesh.Q*mesh.nPoints*sizeof(cuscalar) );

    hipMemcpy( deviceS, S, mesh.Q*mesh.nPoints*sizeof(cuscalar), hipMemcpyHostToDevice );	



    // Factores de relajacion para colision

//    exampleModelCoeffs relax;

    momentoModelCoeffs relax;

    for( uint i = 0 ; i < 9 ; i++ )
	relax.Tau[i] = 1;


    
    cuscalar* deviceTau;

    hipMalloc( (void**)&deviceTau, 9*sizeof(cuscalar) );

    hipMemcpy( deviceTau, relax.Tau, 9*sizeof(cuscalar), hipMemcpyHostToDevice );    


    cuscalar delta_t_cu = 1.0;
    

    // Reduccion

    printf(" Colisión en %d iteraciones\n\n", nit);

    timeInfo Time;

    startTime(&Time);

    for( uint k = 0 ; k < nit ; k++ ) {
	
//    	cudaExampleCollision<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceField, deviceRho, deviceU, deviceTau, cmesh.lattice.M, cmesh.lattice.invM, cmesh.nPoints, cmesh.Q );

	cudaMomentoCollision<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceField, deviceRho, deviceU, deviceF, deviceFint, deviceT, deviceTau, cmesh.lattice.M, cmesh.lattice.invM, cmesh.nPoints, cmesh.Q, delta_t_cu, a, b, c, cs_2, G, sigma, deviceP_EOS, devicePsi, deviceS );

   	hipDeviceSynchronize();

/*      cudaFuerzaFuerzaint<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceFint, devicePsi, cmesh.nPoints, cmesh.Q, cmesh.lattice.vel, cmesh.nb, G);  

    	hipDeviceSynchronize();

        cudaFuerzaFuerzatotal<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceF, deviceFint, deviceRho, g, cmesh.nPoints);	

    	hipDeviceSynchronize();

        cudaMomentoVelocity<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>(deviceField, deviceRho, deviceU, cmesh.lattice.vel, cmesh.nPoints, cmesh.Q, delta_t_cu, deviceF );

    	hipDeviceSynchronize();

        cudaMomentoDensity<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceField, deviceRho, cmesh.nPoints, cmesh.Q);  

    	hipDeviceSynchronize();

        cudaFuerzaS<<<ceil(mesh.nPoints/xgrid)+1,xgrid>>>( deviceS, deviceF, deviceFint, deviceU, devicePsi, sigma, deviceTau[1], deviceTau[2], delta_t_cu,cmesh.nPoints, cmesh.Q);

    	hipDeviceSynchronize();*/

	
    }

    scalar delta_t = 1.0;	
    scalar elap = elapsedTime(&Time);
    
    printf( "\n   Colisión finalizada en %f segundos\n", elap );
    printf( "\n   Tiempo promedio de ejecución por iteración: %g segundos\n\n", elap/nit );
    


    
    // Resultados vuelta al host
    
    cuscalar* dcol = (cuscalar*)malloc( fsize * sizeof(cuscalar) );

    hipMemcpy( dcol, deviceField, fsize*sizeof(cuscalar), hipMemcpyDeviceToHost );

/*-----------------------------------------------------------------------------------------*/
/*
    cuscalar* dP = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) );

    hipMemcpy( dP, deviceP, mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 


    cuscalar* dRho = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) );

    hipMemcpy( dRho, deviceRho, mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 


    cuscalar* dU = (cuscalar*)malloc( 3 * mesh.nPoints * sizeof(cuscalar) );

    hipMemcpy( dU, deviceU, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 


    cuscalar* dT = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); 

    hipMemcpy( dT, deviceT, mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 


    cuscalar* dPsi = (cuscalar*)malloc( mesh.nPoints * sizeof(cuscalar) ); 

    hipMemcpy( dPsi, devicePsi, mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 


    cuscalar* dFint = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) );

    hipMemcpy( dFint, deviceFint, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 


    cuscalar* dF = (cuscalar*)malloc( mesh.nPoints * 3 * sizeof(cuscalar) );

    hipMemcpy( dF, deviceF, 3*mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 


    cuscalar* dS = (cuscalar*)malloc( mesh.nPoints * mesh.Q * sizeof(cuscalar) ); 

    hipMemcpy( dS, deviceS, mesh.Q*mesh.nPoints*sizeof(cuscalar), hipMemcpyDeviceToHost ); 
*/
   
/*-----------------------------------------------------------------------------------------*/



    // Verificacion de calculo contra version de CPU

//    exampleCollision( &mesh, &relax, field, rho, U );

    momentoCollision( &mesh, &relax, field, rho, U, f, fint, T, delta_t, a, b, c, cs_2, G, sigma);
//    momentoCollision( &mesh, &relax, field, rho, U, delta_t, S ); // Calculo de la funcion de distribucion con valores de los parametros seteados para inicializar

							      // A continuacion se calculan el resto de los parametroz para ir actualizandolos
    

/*    fuerzaPsi(psi, p, rho, c, cs_2, G, &mesh);

    fuerzaFuerzaint(fint, psi, &mesh, G);

    fuerzaFuerzatotal(f, fint, rho, g, &mesh);

    momentoVelocity( rho,  U, field, &mesh, delta_t, f);

    momentoDensity( rho, field, &mesh);  

    fuerzaS(S, f, fint, U, psi, sigma, relax.Tau, &mesh, delta_t);
*/

    {
	
    	uint eq = 0;

    	for(uint i = 0 ; i < fsize ; i++) {

	    printf( "%f \t %f \n", dcol[i],field[i]);
//	    printf( "%d \n", eq);	
//	    printf( "%f \n", S[i]);	

    	    if(dcol[i] != field[i])
    		eq = 1;

    	}

    	if(eq != 0)
    	    printf( " Los resultados de host y device difieren!\n " );
    
    }

    
    // Limpieza de memoria

    free( field );

    free( rho );

    free( U ); 

    free( T );

    free( p_EOS );   

    free( psi );

    free( S );

    free( f );   

    free( fint );

    freeBasicMesh( &mesh );

    hipFree( deviceField );

    hipFree( deviceRho );

    hipFree( deviceU );
    
    
    return 0;

}
