#include "hip/hip_runtime.h"
#include <nbReduction.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void nbReduction(cuscalar* field, cuscalar* sum, int* nb, uint np, uint Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   
    if( idx < np ) {

    	uint j;

	cuscalar nodesum = 0;

    	for( j = 0 ; j < Q ; j++ ) {

	    cuscalar nbsum = 0;

	    int nbid = nb[idx*Q + j];
	    
	    if( nbid != -1 ) {

		for( uint l = 0 ; l < Q ; l++ ) {

		    nbsum += field[ nbid*Q + l  ];

		}

	    }

	    nodesum += nbsum;

    	}


    	sum[idx] = nodesum;
	
    }

}
