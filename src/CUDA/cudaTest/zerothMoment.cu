#include <zerothMoment.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



__host__ __device__ void zerothMoment(cuscalar* field, cuscalar* zeroth, basicMesh* mesh ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;

    if( idx <  mesh.nPoints) {

	uint j;

	cuscalar sum = 0;

	for( j = 0 ; j < mesh.Q ; j++ ) {

	    sum += field[ idx*mesh.Q + j ];

	}


	zeroth[idx] = sum;

    }

}
