#include "hip/hip_runtime.h"
#include <zerothMoment.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void zerothMoment(cuscalar* field, cuscalar* zeroth, uint np, uint Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   
    if( idx < np) {

    	uint j;

    	cuscalar sum = 0;

    	for( j = 0 ; j < Q ; j++ ) {

    	    sum += field[ idx*Q + j ];

    	}


    	zeroth[idx] = sum;
	
    }

}
