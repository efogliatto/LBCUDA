#include "hip/hip_runtime.h"
#include <zerothMoment.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



__global__ void zerothMoment(cuscalar* field, cuscalar* zeroth, uint npoints, uint Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;

    if( idx < npoints ) {

    	uint j;

    	cuscalar sum = 0;

    	for( j = 0 ; j < Q ; j++ ) {

    	    sum += field[ idx*Q + j ];

    	}


    	zeroth[idx] = sum;
	
    }

}
