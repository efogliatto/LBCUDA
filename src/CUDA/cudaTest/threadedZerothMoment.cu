#include "hip/hip_runtime.h"
#include <threadedZerothMoment.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void threadedZerothMoment(cuscalar* field, cuscalar* zeroth, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
  
    while( idx < np ) {

    	int j, k = idx*Q;

    	cuscalar sum = 0;

    	for( j = 0 ; j < Q ; j++ ) {

    	    sum += field[ k + j ];

    	}


    	zeroth[idx] = sum;


	// Increase thread index

	idx += blockDim.x * gridDim.x;	
	
    }

}
