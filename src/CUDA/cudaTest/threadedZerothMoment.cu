#include "hip/hip_runtime.h"
#include <threadedZerothMoment.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void threadedZerothMoment(cuscalar* field, cuscalar* zeroth, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   
    while( idx < np ) {

    	int j;

    	cuscalar sum = 0;

    	for( j = 0 ; j < Q ; j++ ) {

    	    sum += field[ idx*Q + j ];

    	}


    	zeroth[idx] = sum;


	// Increade thread index

	idx += blockDim.x * gridDim.x;	
	
    }

}
