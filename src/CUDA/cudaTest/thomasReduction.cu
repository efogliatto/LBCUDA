#include "hip/hip_runtime.h"
#include <thomasReduction.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>


extern "C" __global__ void thomasReduction(cuscalar* field, cuscalar* sum, int* nb, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
 
    int idy = threadIdx.y + blockIdx.y*blockDim.y;
  
    int idz = threadIdx.z + blockIdx.z*blockDim.z;

    if( idx < np ) {



	cuscalar nodesum = 0;

    	if ( idy < Q  ) {

	    cuscalar nbsum = 0;

	    int nbid = nb[idx*Q + idy];


	    
	    if( nbid != -1 ) {
		

		if( idz < Q ) {

		    nbsum += field[ nbid*Q + idz  ];
 

		}

	    }

	    nodesum += nbsum;

    	}


    	sum[idx] = nodesum;
	
    }

}

