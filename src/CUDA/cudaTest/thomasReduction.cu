#include "hip/hip_runtime.h"
#include <thomasReduction.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>


extern "C" __global__ void thomasReduction(cuscalar* field, cuscalar* sum, int* nb, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   
    if( idx < np ) {

    	uint j=0;

	cuscalar nodesum = 0;

    	while( j < Q ) {

	    cuscalar nbsum = 0;

	    int nbid = nb[idx*Q + j];

	    j++;
	    
	    if( nbid != -1 ) {

		uint l = 0 ;

		while(  l < Q ) {

		    nbsum += field[ nbid*Q + l  ];
		    
		    l++;
		}

	    }

	    nodesum += nbsum;

    	}


    	sum[idx] = nodesum;
	
    }

}

