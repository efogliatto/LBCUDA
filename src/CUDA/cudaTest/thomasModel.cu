#include "hip/hip_runtime.h"
#include <thomasModel.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void thomasModel(cuscalar* field, cuscalar* zeroth, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    int idz = threadIdx.z + blockIdx.z * blockDim.z;

	
   // int j=0;	
   
    if( idx < np ) {
  	

    	cuscalar sum = 0;
	if( idy < Q ) {
    		if( idz < Q ) {

    	    	sum += field[ idx*Q + idz ];

		}

    	}


    	zeroth[idx] = sum;
	
    }

}
