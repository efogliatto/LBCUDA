#include "hip/hip_runtime.h"
#include <thomasModel.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void thomasModel(cuscalar* field, cuscalar* zeroth, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    	
   // int j=0;	
   
    if( idx < np ) {
  	

    	cuscalar sum = 0;
	if( idy < Q ) {
    		
    	    	sum += field[ idx*Q + idy ];

		

    	}


    	zeroth[idx] = sum;
	
    }

}
