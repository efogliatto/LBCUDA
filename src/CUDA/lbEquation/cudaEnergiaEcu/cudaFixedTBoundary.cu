#include "hip/hip_runtime.h"
#include <cudaFixedTBoundary.h>

#include <cudaEnergyEqDistNode.h>

#include <stdio.h>

extern "C" __global__ void cudaFixedTBoundary( cuscalar* field,
					       cuscalar* T,
					       cuscalar* U,
					       int* bdPoints,
					       int* nb,
					       cuscalar* invM,
					       cuscalar alpha_1,
					       cuscalar alpha_2,
					       cuscalar bdval,
					       uint bid,
                                               uint nbd,
					       uint maxNbd,
					       uint Q ) {

    
    // Thread index
    
    int idx = threadIdx.x + blockIdx.x*blockDim.x;


    // Calcular solo si el nodo pertenece a esta frontera
    
    if( ( idx >= (bid * maxNbd) )  &&  ( idx < (bid * maxNbd  + maxNbd) )  ) {



	// Lattice constants
    
	cuscalar f_eq_bnd[9];



	// Indice de nodo sobre frontera
	
	int id = bdPoints[idx];

	if( id != -1 ) {



	    // Distribucion de equilibrio sobre la frontera

	    cudaEnergyEqDistNode( f_eq_bnd, T, U, id, Q, invM, alpha_1, alpha_2 );	    
	


	    // Update unknowk distributions

	    for( uint k = 1 ; k < Q ; k++ ) {
			
	        if( nb[id*Q+k] == -1 ) {

		    field[id*Q+k] = f_eq_bnd[k];

	        }

	    }





	    // Correction constants

	    scalar beta=0, kn=0, unk=0;

	    for( uint k = 0 ; k < Q ; k++ ) {
			
	        if( nb[id*Q+k] == -1 ) {

		    unk += field[id*Q+k];

	        }

	        else {

		    kn += field[id*Q+k];
		
	        }

	    }


	    beta = (bdval - kn) / unk;

	    for( uint k = 0 ; k < Q ; k++ ) {
			
	        if( nb[id*Q+k] == -1 ) {

		    field[id*Q+k] = beta * field[id*Q+k];

	        }

	    }

	    

	}







	

    }
    

}
