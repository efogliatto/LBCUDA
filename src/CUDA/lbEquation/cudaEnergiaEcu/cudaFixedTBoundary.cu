#include "hip/hip_runtime.h"
#include <cudaFixedTBoundary.h>

#include <cudaEnergyEqDistNode.h>

#include <stdio.h>

extern "C" __global__ void cudaFixedTBoundary( cuscalar* field,
					       cuscalar* T,
					       cuscalar* U,
					       int* bdPoints,
					       int* nb,
					       cuscalar* invM,
					       cuscalar alpha_1,
					       cuscalar alpha_2,
					       cuscalar bdval,
					       uint bid,
                                               uint nbd,
					       uint maxNbd,
					       uint Q ) {

    
    // Thread index
    
    int idx = threadIdx.x + blockIdx.x*blockDim.x;


    // Calcular solo si el nodo pertenece a esta frontera
    
    if( ( idx >= (bid * maxNbd) )  &&  ( idx < (bid * maxNbd  + maxNbd) )  ) {



	// Lattice constants
    
	cuscalar f_eq_bnd[9];



	// Indice de nodo sobre frontera
	
	int id = bdPoints[idx];

	if( id != -1 ) {

		uint k;

	    // Distribucion de equilibrio sobre la frontera

	    cudaEnergyEqDistNode( f_eq_bnd, T, U, id, Q, invM, alpha_1, alpha_2 );	    
	


		// Update unknowk distributions
		
		k = 1;

	    while( k < Q ) {
			
	        if( nb[id*Q+k] == -1 ) {

		    field[id*Q+k] = f_eq_bnd[k];

	        }

			k++;
	    }





	    // Correction constants

	    scalar beta=0, kn=0, unk=0;
		
		k = 0;

	    while( k < Q ) {
			
	        if( nb[id*Q+k] == -1 ) {

		    unk += field[id*Q+k];

	        }

	        else {

		    kn += field[id*Q+k];
		
	        }
			k++;
	    }


	    beta = (bdval - kn) / unk;
		
		k = 0;

	    while( k < Q ) {
			
	        if( nb[id*Q+k] == -1 ) {

		    field[id*Q+k] = beta * field[id*Q+k];

	        }
			k++;
	    }

	    

	}







	

    }
    

}
