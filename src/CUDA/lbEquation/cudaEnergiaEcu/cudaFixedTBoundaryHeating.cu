#include "hip/hip_runtime.h"
#include <cudaFixedTBoundaryHeating.h>

#include <cudaEnergyEqDistNode.h>

#include <stdio.h>

extern "C" __global__ void cudaFixedTBoundaryHeating( cuscalar* field,
						      cuscalar* T,
						      cuscalar* U,
						      int* bdPoints,
						      int* nb,
						      cuscalar* invM,
						      cuscalar alpha_1,
						      cuscalar alpha_2,
						      cuscalar bdval,
						      cuscalar bdheat,
						      uint widthbd,
						      uint bid,
						      uint nbd,
						      uint maxNbd,
						      uint Q ) {

    
    // Thread index
	
        
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
	
	uint i = (bid * maxNbd);
	uint aux = 0;

	while( i < (bid * maxNbd  + maxNbd)){
		
		if(bdPoints[i]==-1){

			aux = i;
			
			i=(bid * maxNbd  + maxNbd);
		}

		i++;
	}

	
    uint since = (((maxNbd*bid)+aux)/2.0) + 1 - (widthbd/2.0);
    uint until = since + widthbd ;



    // Calcular solo si el nodo pertenece a esta frontera
    
    if( ( idx >= (bid * maxNbd) )  &&  ( idx < (bid * maxNbd  + maxNbd) )  ) {



    	// Lattice constants
		
    	cuscalar f_eq_bnd[9];



    	// Indice de nodo sobre frontera
		
    	int id = bdPoints[idx];

    	if( id != -1 ) {

			uint k;

    		// Distribucion de equilibrio sobre la frontera

    		cudaEnergyEqDistNode( f_eq_bnd, T, U, id, Q, invM, alpha_1, alpha_2 );	    
		


    		// Update unknowk distributions

			k = 1;

			while( k < Q ) {
				
				if( nb[id*Q+k] == -1 ) {
	
				field[id*Q+k] = f_eq_bnd[k];
	
				}
	
				k++;
			}





    		// Correction constants

    		scalar beta=0, kn=0, unk=0;

			while( k < Q ) {
			
				if( nb[id*Q+k] == -1 ) {
	
				unk += field[id*Q+k];
	
				}
	
				else {
	
				kn += field[id*Q+k];
			
				}
				k++;
			}


    		if( idx>=since && idx<=until){

    			beta = (bdheat - kn) / unk;
		
    		}
				
    		else{
		
    			beta = (bdval - kn) / unk;
			
    		}

			k = 0;

			while( k < Q ) {
				
				if( nb[id*Q+k] == -1 ) {
	
				field[id*Q+k] = beta * field[id*Q+k];
	
				}
				k++;
			}

			

    	}







		

    }
    

}
