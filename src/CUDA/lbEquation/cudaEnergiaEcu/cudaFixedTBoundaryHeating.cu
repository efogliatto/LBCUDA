#include "hip/hip_runtime.h"
#include <cudaFixedTBoundaryHeating.h>

#include <cudaEnergyEqDistNode.h>

#include <stdio.h>

extern "C" __global__ void cudaFixedTBoundaryHeating( cuscalar* field,
					       cuscalar* T,
					       cuscalar* U,
					       int* bdPoints,
					       int* nb,
					       cuscalar* invM,
					       cuscalar alpha_1,
					       cuscalar alpha_2,
						   cuscalar bdval,
						   cuscalar bdheat,
						   uint widthbd,
					       uint bid,
                           uint nbd,
					       uint maxNbd,
					       uint Q ) {

    
	// Thread index
	
	printf("widthbd : %d\n",widthbd);
	printf("bid : %d\n",bid);
	printf("nbd : %d\n",nbd);
	printf("maxNbd : %d\n",maxNbd);
    							
    
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	
	
	uint since = (nbd/2.0)+1 - (widthbd/2.0);
	uint until = since + widthbd ;



    // Calcular solo si el nodo pertenece a esta frontera
    
    if( ( idx >= (bid * maxNbd) )  &&  ( idx < (bid * maxNbd  + maxNbd) )  ) {



		// Lattice constants
		
		cuscalar f_eq_bnd[9];



		// Indice de nodo sobre frontera
		
		int id = bdPoints[idx];

		if( id != -1 ) {



			// Distribucion de equilibrio sobre la frontera

			cudaEnergyEqDistNode( f_eq_bnd, T, U, id, Q, invM, alpha_1, alpha_2 );	    
		


			// Update unknowk distributions

			for( uint k = 1 ; k < Q ; k++ ) {
				
				if( nb[id*Q+k] == -1 ) {

					field[id*Q+k] = f_eq_bnd[k];

				}

			}





			// Correction constants

			scalar beta=0, kn=0, unk=0;

			for( uint k = 0 ; k < Q ; k++ ) {
				
				if( nb[id*Q+k] == -1 ) {

					unk += field[id*Q+k];

				}

				else {

					kn += field[id*Q+k];
			
				}

			}


			if( i>=since && i<=until){

				beta = (bdheat - kn) / unk;
		
			}
				
			else{
		
				beta = (bdval - kn) / unk;
			
			}

			for( uint k = 0 ; k < Q ; k++ ) {
				
				if( nb[id*Q+k] == -1 ) {

					field[id*Q+k] = beta * field[id*Q+k];

				}

			}

			

		}







		

    }
    

}
