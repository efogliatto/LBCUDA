#include "hip/hip_runtime.h"
#include <cudaEnergyS.h>

#include <cudaEnergy.h>

#include <cudaEnergyGradient.h>

#include <cudaEnergyGradientU.h>

#include <stdlib.h>

#include <stdio.h>

extern "C" __global__ void cudaEnergyS( cuscalar* s, cuscalar* rho, cuscalar* T, cuscalar* U, cudaEnergyCoeffs* relax, cuscalar* field, cuscalar alpha_1, cuscalar alpha_2,  cuscalar cs_2, cuscalar delta_t, cuscalar c_v, cuscalar b, unit np, int Q, int* lvel,int* nb, cuscalar* Tau) {

    int id = threadIdx.x + blockIdx.x*blockDim.x;

        cuscalar chi = delta_t * ( ( 1 / Tau[2]) -0.5 ) * ( (4 + 3 *alha_1 +2 * alpha_2 ) / 6.0 );
        
        cuscalar grad_Rho[3];
        cuscalar grad_T[3];
        cuscalar grad_Ux[3];
        cuscalar grad_Uy[3];
        cuscalar grad_Uz[3]={0,0,0};


        if( id > np ){

            cuscalar auxField[9] = {0,0,0,0,0,0,0,0,0};
		    
		    // Copia de parametros auxiliares para el calculo de S
            uint k = 0 ;			    
		    
            while( k < 9 )	{
					
			    auxField[k] = field[id*9 + k];

                k++;
		    }
            
            cudaEnergyGradient( grad_T, auxField, T, id, cs_2, delta_t, Q, lvel, nb);
            cudaEnergyGradient( grad_Rho, auxField, rho, id, cs_2, delta_t, Q, lvel, nb);
            cudaEnergyGradientU( grad_Ux, auxField, 0, U, id, cs_2, delta_t, Q, lvel, nb);
            cudaEnergyGradientU( grad_Uy, auxField, 1, U, id, cs_2, delta_t, Q, lvel, nb);
            //cudaEnergyGradientU( grad_Uz, auxField, 2, U, id, cs_2, delta_t, Q, lvel, nb); debido a que para D2Q9 grad_Uz = 0

            s[id] = (chi/rho[id])*(grad_T[0]*grad_Rho[0]+grad_T[1]*grad_Rho[1]+grad_T[2]*grad_Rho[2]) + T[id]*(1-(1/(rho[id]*c_v)*(rho[id]*R/(1-rho[id]*b))))*(grad_Ux[0]+grad_Uy[1]+grad_Uz[2]);

        }
}

