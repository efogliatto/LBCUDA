#include "hip/hip_runtime.h"
#include <cudaEnergySource.h>

#include <cudaFDOperations.h>

#include <stdlib.h>

#include <stdio.h>

extern "C" __global__ void cudaEnergySource( cuscalar* s,
					     cuscalar* rho,
					     cuscalar* T,
					     cuscalar* U,
					     cuscalar* Tau,
					     cuscalar alpha_1,
					     cuscalar alpha_2,
					     cuscalar cs_2,
					     cuscalar Cv,
					     cuscalar b,
					     uint np,
					     uint Q,
					     int* lvel,
					     int* nb ) {

    
    // Thread index
    
    int id = threadIdx.x + blockIdx.x*blockDim.x;


    if( id < np ) {



	// Cached scalar values

	cuscalar _rho = rho[id];

	cuscalar _T = T[id];
	
	
    
	// Thermal difusivity
    
	cuscalar chi = (1/Tau[3] - 0.5) * (4.0 + 3.0 * alpha_1  + 2.0 * alpha_2) / 6.0;
	


	// Scalar fields gradients

	cuscalar gradT[3]   = {0,0,0};

	cuscalar gradRho[3] = {0,0,0};
	
	cudaScalarGradient( gradT, T, nb, id, Q );

	cudaScalarGradient( gradRho, rho, nb, id, Q );


	
	cuscalar first = 0;
	uint j = 0;

	while( j < 3 ){
	    first += gradT[j] * gradRho[j];
		j++;
	}
	
	first = first * chi / _rho;

	
	
	// Velocity divergence term

	cuscalar dp_dT = _rho / (1 - _rho * b);

	cuscalar divU;
	cudaVectorDivergence( &divU, U, nb, id, Q );

	cuscalar second = divU * _T * ( 1.0   -   dp_dT / (_rho * Cv) );


	
	// Update source at node
	
	s[id] = first + second;
	

    }

}

