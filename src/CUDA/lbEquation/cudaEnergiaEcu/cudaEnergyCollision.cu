#include "hip/hip_runtime.h"
#include <cudaEnergyCollision.h>

#include <stdlib.h>

#include <stdio.h>



extern "C" __global__ void cudaEnergyCollision( cuscalar* field,
						cuscalar* U,
						cuscalar* T,
						cuscalar* gamma_0,
                                                cuscalar* Tau,
						cuscalar  alpha_1,
						cuscalar  alpha_2,
						uint Q,
						uint np,
						cuscalar* M,
						cuscalar* invM ) {


    // Thread index
    
    int id = threadIdx.x + blockIdx.x*blockDim.x;



	if( id < np ) {
	
	
		// Distribuciones parciales
		
		cuscalar n[9];   //n:  Distribucion en espacio de momentos
		
		cuscalar n_eq[9];   // neq: Distribucion de equilibrio en espacio de momentos

		cuscalar aux[9];

		cuscalar Q_34 = Tau[4] *(Tau[3]*0.5 -1);

		cuscalar Q_56 = Tau[6] *(Tau[5]*0.5 -1);



	

		scalar ux = U[id*3], uy = U[id*3 + 1];

		scalar T_ = T[id];
		
		
		// Distribucion de equilibrio en espacio de momentos

		n_eq[0] = T_;
		n_eq[1] = alpha_1 * T_;       
		n_eq[2] = alpha_2 * T_;	
		n_eq[3] = T_ * ux;	
		n_eq[4] = T_ * (-ux);	
		n_eq[5] = T_ * uy;	
		n_eq[6] = T_ * (-uy);	
		n_eq[7] = 0;	
		n_eq[8] = 0;

	
		
		// Distribucion en espacio de momentos. n = M*field[id]
		
		uint i = 0;
		
		while( i < Q ) {

			n[i] = 0;

			uint j = 0;

			while( j < Q ) { 

				n[i] += M[i*Q + j] * field[ id*Q + j ];
			
				j++;
			}

			i++;
		}



		// Distribucion auxiliar: diag(Q) * ( n - n_eq )

		uint k = 0;

		while( k < Q ){
			
			aux[k] = Tau[k] * ( n[k] - n_eq[k] );
			
			k++;
		}

		// Correccion

		aux[3] = aux[3] + Q_34*( n[4] - n_eq[4] );

		aux[5] = aux[5] + Q_56*( n[6] - n_eq[6] );



		// Second auxiliary distribution: (I  -  0.5 * Q) * GammaHat	

		scalar heat = (1.0 - 0.5 * Tau[0])  * gamma_0[id];


	
		// Completa la colision

		k = 0;
	
		while( k < Q ){
			
			n[k] = n[k] - aux[k];
			
			k++;

		}
	
		n[0] += heat;






	
		//-------------------------------------------------
		
		// Vuelta al espacio de fases. field = invM * n

		i = 0;

		while( i < Q ) {

			field[id*Q + i] = 0;

			uint j = 0;

			while( j < Q ) {

				field[id*Q + i] = field[id*Q + i] + invM[i*Q + j] * n[j];
				
				j++;
			}
			
			i++;
		}



	}
	
}
