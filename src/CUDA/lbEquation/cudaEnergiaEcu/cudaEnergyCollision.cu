#include "hip/hip_runtime.h"
#include <cudaEnergyCollision.h>

#include <stdlib.h>

#include <stdio.h>



extern "C" __global__ void cudaEnergyCollision( cuscalar* field, cuscalar* gamma_0,  cudaEnergyCoeffs* relax, cuscalar delta_t, unit np, int Q, int* lvel,int* nb, cuscalar* Tau, cuscalar* M, cuscalar* invM) {
    
	int id = threadIdx.x + blockIdx.x*blockDim.x;

    // Distribuciones parciales
    
    cuscalar n[9];   // n:  Distribucion en espacio de momentos
    
    cuscalar n_eq[9];   // neq: Distribucion de equilibrio en espacio de momentos

	cuscalar Q_34 = Tau[3] *(Tau[2]*0.5 -1); // Valor de la matriz de relajación Q

	cuscalar Q_56 = Tau[5] *(Tau[4]*0.5 -1); // Valor de la matriz de relajación Q
    
    // Colision sobre todos los nodos
    
    if( id < np ) {
		
		uint i = 0;
		uint j = 0;
		uint k = 0;

		// Distribucion de equilibrio en espacio de momentos
		
		n_eq[0] = T[id];
		n_eq[1] = T[id] * field[id*Q];
		n_eq[2] = T[id] * field[id*Q+1];
		n_eq[3] = T[id] * ux;
		n_eq[4] = T[id] * (-ux);
		n_eq[5] = T[id] * uy;
		n_eq[6] = T[id] * (-uy);
		n_eq[7] = 0;
		n_eq[8] = 0;
		
		// Distribucion en espacio de momentos. n = M*field[id]
											
		while( i < Q ) {

			n[i] = 0;

			while( j < Q ) { 

				n[i] += M[i*Q + j] * field[ id*Q + j ];

				j++;
			}

			i++;
		}

 			// Collision in momentum space//--------------------------------
		
		while( k < Q ){
			if ( k == 0){
				n[k] = ( n[k]  -  Tau[k]*( n[k] - n_eq[k] ) ) + ( delta_t * ( 1 - 0.5 * Tau[k] ) * gamma_0[id] );
			}


			if ( k == 2){
				n[k] = ( n[k]  -  (Tau[k]*( n[k] - n_eq[k] ) + Q_34 *( n[3] - n_eq[3] ) ) );
			}
			
			if ( k == 4){
				n[k] = ( n[k]  -  (Tau[k]*( n[k] - n_eq[k] ) + Q_56 *( n[5] - n_eq[5] ) ) );
			}

			else{
				n[k] = ( n[k]  -  Tau[k]*( n[k] - n_eq[k] ) );
			}
						
			k++
		}
			//-------------------------------------------------
		
		// Vuelta al espacio de fases. field = invM * n

		i=0;
		j=0;

		while( i < Q ) {

			field[id*Q + i] = 0;

			while( j < Q ) {

				cuscalar aux = invM[i*Q + j] * m[j];

				field[id*Q + i] = field[id*Q + i] + aux;
				
				j++;
			}
			
			i++;
		}
		
    }
	
}
