#include "hip/hip_runtime.h"
#include <cudaEnergyTemp.h>

#include <stdio.h>

extern "C" __global__ void cudaEnergyTemp( cuscalar* T, cuscalar* field, cuscalar* gamma_0, uint np, uint Q) {

    int id = threadIdx.x + blockIdx.x*blockDim.x;

    
    // Suma de todas las componentes
    
    if( id < np ) {

	cuscalar acum = 0;	    
	uint j = 0 ;

	while( j < Q ) {

	    acum += field[ id*Q + j ];

		j++;
	}

	T[id] = acum + 0.5*gamma_0[id];	

    }
    

}
