#include "hip/hip_runtime.h"
#include <cudaEnergyTemp.h>

#include <stdio.h>

extern "C" __global__ void cudaEnergyTemp( cuscalar* T, cuscalar* field, cuscalar* gamma_0, cuscalar delta_t, unit np, int Q) {

    int id = threadIdx.x + blockIdx.x*blockDim.x;

    // Suma de todas las componentes
    
    if( id < np ) {

		T[id] = 0;	    

		uint j = 0;

		while( j < Q ) {

	    	T[id] += field[ id*Q + j ];

			j++;
		}

		T[id]+= delta_t * gamma_0[id]

    }
    

}
