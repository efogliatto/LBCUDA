#include "hip/hip_runtime.h"
#include <cudaEnergyTemp.h>

#include <stdio.h>

extern "C" __global__ void cudaEnergyTemp( cuscalar* T, cuscalar* field, cuscalar* gamma_0, uint np, uint Q) {

    int id = threadIdx.x + blockIdx.x*blockDim.x;

    
    // Suma de todas las componentes
    
    if( id < np ) {

	cuscalar acum = 0;	    

	for( uint j = 0 ; j < Q ; j++ ) {

	    acum += field[ id*Q + j ];

	}

	T[id] = acum + 0.5*gamma_0[id];	

    }
    

}
