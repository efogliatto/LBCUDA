#include "hip/hip_runtime.h"
#include <cudaSwap.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaSwap(cuscalar* field, cuscalar* swap, int* nb, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   
    if( idx < np ) {

	for( uint k = 0 ; k < Q ; k++ ) {

	    field[idx*Q+k] = swap[idx*Q+k];
	    
	}
	
    }

}
