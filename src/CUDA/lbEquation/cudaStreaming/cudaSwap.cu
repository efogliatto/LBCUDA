#include "hip/hip_runtime.h"
#include <cudaSwap.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaSwap(cuscalar* field, cuscalar* swap, int* nb, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   
    if( idx < np ) {

        uint k = 0;

	while( k < Q ) {

	    field[idx*Q+k] = swap[idx*Q+k];
	    k++;
	}
	
    }

}
