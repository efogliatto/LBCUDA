#include "hip/hip_runtime.h"
#include <cudaStream.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaStream(cuscalar* field, cuscalar* swap, int* nb, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   
    if( idx < np ) {

	
	// Copy only neighbours to swap   
	uint k = 0;

	for( k < Q ) {

	    int neighId = nb[idx*Q+k];

	    if( neighId != -1 ) {

		swap[idx*Q+k] = field[neighId*Q+k];

	    }

	    else {

		swap[idx*Q+k] = field[idx*Q+k];

	    }
    	k++;
	}

	
    }

}
