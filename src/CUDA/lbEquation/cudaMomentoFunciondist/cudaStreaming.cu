#include "hip/hip_runtime.h"
#include <cudaStreaming.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaStreaming(cuscalar* field, cuscalar* swap, int* nb, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   
    if( idx < np ) {

	
	// Copy only neighbours to swap   

	for( uint i = 0 ; i < np ; i++ ) {

	    for( uint k = 0 ; k < Q ; k++ ) {

		int neighId = nb[i*Q+k];

		if( neighId != -1 ) {

		    swap[i*Q+k] = field[neighId*Q+k];

		}

		else {

		    swap[i*Q+k] = field[i*Q+k];

		}
    
	    }

	}



	// Copy back from swap
    
	for( uint i = 0 ; i < np ; i++ ) {

	    for( uint k = 0 ; k < Q ; k++ ) {

		field[i*Q+k] = swap[i*Q+k];
	    
	    }

	}
	
    }

}
