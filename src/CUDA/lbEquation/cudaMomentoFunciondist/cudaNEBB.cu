#include "hip/hip_runtime.h"
#include <cudaNEBB.h>

#include <cudaEnergyEqDistNode.h>

#include <stdio.h>

extern "C" __global__ void cudaNEBB( cuscalar* field,
				     cuscalar* Ft,
				     int* bdPoints,
				     uint bid,
				     uint bndNormal,
				     uint maxNbd,
				     uint Q ) {

    
    // Thread index
    
    int idx = threadIdx.x + blockIdx.x*blockDim.x;


    // Calcular solo si el nodo pertenece a esta frontera
    
    if( ( idx >= (bid * maxNbd) )  &&  ( idx < (bid * maxNbd  + maxNbd) )  ) {


	// Indice de nodo sobre frontera
	
	int id = bdPoints[idx];

	if( id != -1 ) {


	    // Fuerza total

	    cuscalar Ftotal[3] = {0,0,0};

		uint j=0;
	    while( j < 3 ){
			Ftotal[j] = Ft[ id*3 + j ];
			j++;	
		}

	    switch( bndNormal ) {


	    case 0:  // X0

		field[id*Q+1] = field[id*Q+3];

		field[id*Q+5] = field[id*Q+7] + 0.5 * (  field[id*Q+4] - field[id*Q+2] ) + 0.25*(Ftotal[0]+Ftotal[1]);

		field[id*Q+8] = field[id*Q+6] - 0.5 * (  field[id*Q+4] - field[id*Q+2] ) + 0.25*(Ftotal[0]-Ftotal[1]);

		break;


	    case 1:  // X1

		field[id*Q+3] = field[id*Q+1];

		field[id*Q+7] = field[id*Q+5] - 0.5 * (  field[id*Q+4] - field[id*Q+2] ) - 0.25*(Ftotal[0]+Ftotal[1]);

		field[id*Q+6] = field[id*Q+8] + 0.5 * (  field[id*Q+4] - field[id*Q+2] ) - 0.25*(Ftotal[0]-Ftotal[1]);

		break;


	    case 2:  // Y0

		field[id*Q+2] = field[id*Q+4];

		field[id*Q+5] = field[id*Q+7] - 0.5 * (  field[id*Q+1] - field[id*Q+3] ) - 0.25*(Ftotal[0]+Ftotal[1]);

		field[id*Q+6] = field[id*Q+8] + 0.5 * (  field[id*Q+1] - field[id*Q+3] ) + 0.25*(Ftotal[0]-Ftotal[1]);

		break;


	    case 3:  // Y1

		field[id*Q+4] = field[id*Q+2];

		field[id*Q+7] = field[id*Q+5] + 0.5 * (  field[id*Q+1] - field[id*Q+3] ) + 0.25*(Ftotal[0]+Ftotal[1]);

		field[id*Q+8] = field[id*Q+6] - 0.5 * (  field[id*Q+1] - field[id*Q+3] ) - 0.25*(Ftotal[0]-Ftotal[1]);		

		break;	    
	    

	    }
	    


	}


    }
    

}
