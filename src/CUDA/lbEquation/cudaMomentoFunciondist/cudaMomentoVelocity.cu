#include "hip/hip_runtime.h"
#include <cudaMomentoVelocity.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaMomentoVelocity(cuscalar* field, cuscalar* rho, cuscalar* U, cuscalar* Ft, int* lvel, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
  
   
    if( idx < np ) {

	
	// Velocidad local
	
	cuscalar lv[3] = {0,0,0};


	// Move over velocity components
	
	uint j = 0;

	while( j < 3 ) {

	    
	    // Move over model velocities
	    uint k =0;
	    while( k < Q ){
		lv[j] += (cuscalar)lvel[k*3+j] * field[idx*Q+k];
			k++;
		}
	
	    lv[j] += 0.5 * Ft[ idx * 3 + j ];
		
		j++;
	}


	
	// Add interaction force and divide by density
	
	j=0;

	while( j < 3 ){
	    lv[j] = lv[j] / rho[idx];
		j++;
	}


	// Copy to global array
	j=0;
	while( j < 3 )	{
	    U[idx*3+j] = lv[j];
		j++;
	}
	

    }

}
