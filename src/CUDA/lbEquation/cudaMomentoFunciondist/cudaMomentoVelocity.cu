#include "hip/hip_runtime.h"
#include <cudaMomentoVelocity.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaMomentoVelocity(cuscalar* field, cuscalar* rho, cuscalar* U, cuscalar* Ft, int* lvel, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
  
   
    if( idx < np ) {

	
	// Velocidad local
	
	cuscalar lv[3] = {0,0,0};


	// Move over velocity components
	
	for( uint j = 0 ; j < 3 ; j++ ) {

	    
	    // Move over model velocities
	    
	    for(uint k = 0 ; k < Q ; k++)
		lv[j] += (cuscalar)lvel[k*3+j] * field[idx*Q+k];

	
	    lv[j] += 0.5 * Ft[ idx * 3 + j ];
	
	}


	
	// Add interaction force and divide by density
	
	for( uint j = 0 ; j < 3 ; j++ )
	    lv[j] = lv[j] / rho[idx];
	


	// Copy to global array
	for(uint j = 0 ; j < 3 ; j++)	
	    U[idx*3+j] = lv[j];
	
	

    }

}
