#include "hip/hip_runtime.h"
#include <cudaMomentoVelocity.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaMomentoVelocity(cuscalar* field, cuscalar* rho, cuscalar* U, int* lvel, int np, int Q, cuscalar delta_t, cuscalar* F ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    
   
    if( idx < np ) {


	// Velocidad local
	
	cuscalar lv[3] = {0,0,0};


	// Indice sobre componentes de velocidad
	
	uint j = 0 ;

	while ( j < 3 ) {

	    
	    // Indice sobre componentes de la funcion de distribucion
	    
	    uint k = 0 ;

	    while ( k < Q ) {

		lv[j] += lvel[3*k + j] * field[idx*Q + k];
		    
		k++;

	    }

	    lv[j] += 0.5 * delta_t * F[idx * 3 + j];

	    j++;

	    
	}


	// Divide por densidad
	
	j = 0;

	while ( j < 3 ) {

	    lv[j] = lv[j] / rho[idx];
	
	    j++;

	}


	
	// Copia al arreglo global
	
	j = 0;

	while ( j < 3 ) {
	
	    U[idx*Q + j] = lv[j];
	
	    j++;

	}


	
    }

}
