#include "hip/hip_runtime.h"
#include <cudaMomentoCollision.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>

#include <cudaFuerza.h>


extern "C" __global__ void cudaMomentoCollision( cuscalar* field, cuscalar* rho, cuscalar* U, cuscalar* f,
						 cuscalar* fint, cuscalar* T, cuscalar* Tau, cuscalar* M,
						 cuscalar* invM, uint np, uint Q, cuscalar delta_t_cu,
						 cuscalar a, cuscalar b, cuscalar c, cuscalar cs_2, cuscalar G, cuscalar sigma) {
    

    // Thread index
    
    int id = threadIdx.x + blockIdx.x*blockDim.x;

   
    if( id < np ) {


	// Distribuciones parciales
    
	cuscalar m[9];   // m:  Distribucion en espacio de momentos
    
	cuscalar m_eq[9];   // meq: Distribucion de equilibrio en espacio de momentos


	// Magnitud de la velocidad
		
	cuscalar ux = U[id*3], uy = U[id*3 + 1], uz = U[id*3 + 2];

	cuscalar umag = ux*ux + uy*uy + uz*uz;

	
	// Copia de parametros auxiliares para el calculo de S

	cuscalar auxU[3] = {0,0,0};
	cuscalar auxF[3] = {0,0,0};
	cuscalar auxFint[3] = {0,0,0};
	cuscalar s[9] = {0,0,0,0,0,0,0,0,0};


	for( uint k = 0 ; k < 3 ; k++ )	{
			
	    auxU[k] = U[id*3 + k];
	    auxF[k] = f[id*3 + k];
	    auxFint[k] = fint[id*3 + k];
		
	}

	for( uint k = 0 ; k < Q ; k++ )	{

	    s[k] = 0.0 ;

	}	

		
	// Distribucion de equilibrio en espacio de momentos

	cuscalar localRho = rho[id];
		
	m_eq[0] = localRho;
	m_eq[1] = localRho * (-2 + 3*umag);
	m_eq[2] = localRho * (1 - 3*umag);
	m_eq[3] = localRho * ux;
	m_eq[4] = localRho * (-ux);
	m_eq[5] = localRho * uy;
	m_eq[6] = localRho * (-uy);
	m_eq[7] = localRho * (ux*ux - uy*uy);
	m_eq[8] = localRho * ux * uy;



	// Distribucion en espacio de momentos. m = M*field[id]

	for( uint i = 0 ; i < Q ; i++ ) {

	    m[i] = 0;

	    for( uint j = 0 ; j < Q ; j++ ) {

		m[i] += M[i*Q + j] * field[ id*Q + j ];

	    }

	}
	
	/* // Distribucion en espacio de momentos. m = M*field[id] */

	/* uint i = 0 ; */

	/* while ( i < Q ) { */

	/*     m[i] = 0; */
			
	/*     uint j = 0 ;		 */

	/*     while ( j < Q ) { */

	/* 	m[i] += M[i*Q + j] * field[ id*Q + j ]; */
				

	/* 	j++;	 */
	/*     } */
			

	/*     i++;	 */
	/* } */


	// Calculo de S termino de fuente
	
	cuscalar p_EOS = 0.0;
			
	cuscalar psi = 0.0;

	cudaFuerzaPresionEOS( &p_EOS, rho[id] , T[id], a, b); 

	cudaFuerzaPsi( &psi, p_EOS, rho[id], c, cs_2, G);

	cudaFuerzaS(s, auxF, auxFint, auxU, psi, sigma, Tau, delta_t_cu) ;



	
	// Collision in momentum space

	for( uint k = 0 ; k < Q ; k++ )
	    m[k] = ( m[k]  -  Tau[k]*( m[k] - m_eq[k] ) ) + ( ( 1 - 0.5 * Tau[k] ) * s[k] );
	
	
	/* uint k = 0 ;	 */

	/* while ( k < Q ){ */

	/*     m[k] = ( m[k]  -  Tau[k]*( m[k] - m_eq[k] ) ) + ( delta_t_cu * ( 1 - 0.5 * Tau[k] ) * s[k] ); */
			
	/*     k++;	 */
			
	/* } */


	
		
	// Vuelta al espacio de fases. field = invM * m

	for( uint i = 0 ; i < Q ; i++ ) {

	    field[id*Q + i] = 0;

	    for( uint j = 0 ; j < Q ; j++ ) {

		field[id*Q + i] += invM[i*Q + j] * m[j];

	    }

	}
	
	
	/* i=0; */
	/* while ( i < Q ) { */

	/*     field[id*Q + i] = 0; */

	/*     uint j = 0 ; */

	/*     while ( j < Q ) { */

	/* 	cuscalar aux = invM[i*Q + j] * m[j]; */

	/* 	field[id*Q + i] = field[id*Q + i] + aux; */
			
	/* 	j++;	 */
	
	/*     } */
	/*     i++;		 */
			

	/* } */

	
		
    }

}
