#include "hip/hip_runtime.h"
#include <cudaMomentoCollision.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>

#include <cudaFuerza.h>

#include <cudaMomentoFunciondist.h>

extern "C" __global__ void cudaMomentoCollision( cuscalar* field, cuscalar* rho, cuscalar* U, cuscalar* f, cuscalar* fint, cuscalar* T, cuscalar* Tau, cuscalar* M, cuscalar* invM, uint np, uint Q, cuscalar delta_t, int a, int b, cuscalar c, cuscalar cs_2, cuscalar G, cuscalar sigma, cuscalar* s ) {
    

    int id = threadIdx.x + blockIdx.x*blockDim.x;

   
    if( id < np ) {


	// Distribuciones parciales
    
	cuscalar m[9];   // m:  Distribucion en espacio de momentos
    
	cuscalar m_eq[9];   // meq: Distribucion de equilibrio en espacio de momentos


	// Magnitud de la velocidad
	
	cuscalar ux = U[id*3], uy = U[id*3 + 1], uz = U[id*3 + 2];

	cuscalar umag = ux*ux + uy*uy + uz*uz;
	
	// Copia de parametros auxiliares para el calculo de S


	cuscalar auxU[3] = {0,0,0};
	cuscalar auxF[3] = {0,0,0};
	cuscalar auxFint[3] = {0,0,0};


	for( uint k = 0 ; k < 3 ; k++ )	{
		
		auxU[k] = U[id*3 + k];
		auxF[k] = f[id*3 + k];
		auxFint[k] = fint[id*3 + k];
	
	}

	for( uint k = 0 ; k < Q ; k++ )	{

		s[k] = 0.0 ;

	}	

	
	// Distribucion de equilibrio en espacio de momentos

	cuscalar localRho = rho[id];
	
	m_eq[0] = localRho;
	m_eq[1] = localRho * (-2 + 3*umag);
	m_eq[2] = localRho * (1 - 3*umag);
	m_eq[3] = localRho * ux;
	m_eq[4] = localRho * (-ux);
	m_eq[5] = localRho * uy;
	m_eq[6] = localRho * (-uy);
	m_eq[7] = localRho * (ux*ux - uy*uy);
	m_eq[8] = localRho * ux * uy;


	
	// Distribucion en espacio de momentos. m = M*field[id]

	uint i = 0 ;

	while ( i < Q ) {

	    m[i] = 0;
		
	    uint j = 0 ;		

	    while ( j < Q ) {

		m[i] += M[i*Q + j] * field[ id*Q + j ];

		j++;	

	    }
	
	    i++;	

	}


	cuscalar p_EOS = 0.0;
		
	cuscalar psi = 0.0;

  	printf("p_EOS antes :%f \n ", p_EOS);
/*
	cudaFuerzaPresionEOS( &p_EOS, rho[id] , T[id], a, b); 	
  
	printf("p_EOS despues :%f \n ", p_EOS);

	cudaFuerzaPsi( &psi, p_EOS, rho[id], c, cs_2, G);

	cudaFuerzaS(s, auxF, auxFint, auxU, psi, sigma, Tau, delta_t) ;
*/


	
	/* cudaThreadedMatMult<<<1,9>>>(M, field, id, Q); */

	/* __syncthreads(); */
	

	
	// Collision in momentum space
	
	uint k = 0 ;	

	while ( k < Q ){

	    m[k] = ( m[k]  -  Tau[k]*( m[k] - m_eq[k] ) ) + ( delta_t * ( 1 - 0.5 * Tau[k] ) * s[k] );
	    
	    k++;	

	}
	
	// Vuelta al espacio de fases. field = invM * m

	i=0;

	while ( i < Q ) {

	    field[id*Q + i] = 0;

	    uint j = 0 ;

	    while ( j < Q ) {

		field[id*Q + i] += invM[i*Q + j] * m[j];
		
		j++;		

	    }

	    i++;

	}
	


    }

}
