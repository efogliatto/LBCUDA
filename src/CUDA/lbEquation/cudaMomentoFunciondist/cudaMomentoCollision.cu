#include "hip/hip_runtime.h"
#include <cudaMomentoCollision.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>

#include <cudaThreadedMatMult.h>



extern "C" __global__ void cudaMomentoCollision( cuscalar* field, cuscalar* rho, cuscalar* U, cuscalar* Tau, cuscalar* M, cuscalar* invM, uint np, uint Q, cuscalar delta_t ) {
    

    int id = threadIdx.x + blockIdx.x*blockDim.x;

   
    if( id < np ) {


	// Distribuciones parciales
    
	cuscalar m[9];   // m:  Distribucion en espacio de momentos
    
	cuscalar m_eq[9];   // meq: Distribucion de equilibrio en espacio de momentos

	cuscalar s[9] = {0,0,0,0,0,0,0,0,0};	// s: Termino de fuente que se debe traer ya calculado 


	// Magnitud de la velocidad
	
	cuscalar ux = U[id*3], uy = U[id*3 + 1], uz = U[id*3 + 2];

	cuscalar umag = ux*ux + uy*uy + uz*uz;
	


	
	// Distribucion de equilibrio en espacio de momentos

	cuscalar localRho = rho[id];
	
	m_eq[0] = localRho;
	m_eq[1] = localRho * (-2 + 3*umag);
	m_eq[2] = localRho * (1 - 3*umag);
	m_eq[3] = localRho * ux;
	m_eq[4] = localRho * (-ux);
	m_eq[5] = localRho * uy;
	m_eq[6] = localRho * (-uy);
	m_eq[7] = localRho * (ux*ux - uy*uy);
	m_eq[8] = localRho * ux * uy;


	
	// Distribucion en espacio de momentos. m = M*field[id]

	uint i = 0 ;

	while ( i < Q ) {

	    m[i] = 0;
		
	    uint j = 0 ;		

	    while ( j < Q ) {

		m[i] += M[i*Q + j] * field[ id*Q + j ];

		j++;	

	    }
	
	    i++;	

	}

	
	/* cudaThreadedMatMult<<<1,9>>>(M, field, id, Q); */

	/* __syncthreads(); */
	

	
	// Collision in momentum space
	
	uint k = 0 ;	

	while ( k < Q ){

	    m[k] = ( m[k]  -  Tau[k]*( m[k] - m_eq[k] ) ) + ( delta_t * ( 1 - 0.5 * Tau[k] ) * s[k] );
	    
	    k++;	

	}
	
	// Vuelta al espacio de fases. field = invM * m

	i=0;
	uint j = 0 ;		

	while ( i < Q ) {

	    field[id*Q + i] = 0;

	    while ( j < Q ) {

		field[id*Q + i] += invM[i*Q + j] * m[j];
		
		j++;		

	    }

	    i++;

	}
	


    }

}
