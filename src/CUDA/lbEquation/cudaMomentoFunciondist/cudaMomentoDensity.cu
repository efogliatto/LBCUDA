#include "hip/hip_runtime.h"
#include <cudaMomentoDensity.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaMomentoDensity(cuscalar* field, cuscalar* rho, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   
    if( idx < np ) {

    	int j= 0;

    	cuscalar sum = 0;

    	while ( j < Q ) {

    	    sum += field[ idx*Q + j ];
		
	    j++;	
	
    	}


    	rho[idx] = sum;
	
    }

}
