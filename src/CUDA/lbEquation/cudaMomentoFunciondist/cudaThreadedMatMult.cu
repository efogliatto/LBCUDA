#include "hip/hip_runtime.h"
#include <cudaThreadedMatMult.h>

#include <stdio.h>


extern "C" __global__ void cudaThreadedMatMult(cuscalar* mat, cuscalar* vec, uint id, uint Q) {

    extern __shared__ cuscalar m[];
    
    int thid = threadIdx.x;

    if(thid < Q) {	
    
	m[thid] = 0;

	for( uint j = 0 ; j < Q ; j++ ) {

	    m[thid] += mat[thid*Q + j] * vec[ id*Q + j ];

	}

    }

}
