#include "hip/hip_runtime.h"
#include <cudaExampleCollision.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaExampleCollision(cudaBasicMesh* mesh, cuscalar* field, cuscalar* rho, cuscalar* U ) {

    int id = threadIdx.x + blockIdx.x*blockDim.x;

   
    if( id < mesh->nPoints ) {


	printf("%d %d\n", id, mesh->lattice.q);	
	

	// Distribuciones parciales
    
	cuscalar m[9];   // m:  Distribucion en espacio de momentos
    
	cuscalar m_eq[9];   // meq: Distribucion de equilibrio en espacio de momentos



	// Magnitud de la velocidad
	
	scalar umag = 0;
	
	for( uint k = 0 ; k < 3 ; k++ )	
	    umag += U[id*3 + k] * U[id*3 + k];


	
	// Distribucion de equilibrio en espacio de momentos
	
	m_eq[0] = rho[id];
	m_eq[1] = rho[id] * (-2 + 3*umag);
	m_eq[2] = rho[id] * (1 - 3*umag);
	m_eq[3] = rho[id] * U[id*3];
	m_eq[4] = rho[id] * (-U[id*3]);
	m_eq[5] = rho[id] * U[id*3+1];
	m_eq[6] = rho[id] * (-U[id*3+1]);
	m_eq[7] = rho[id] * (U[id*3]*U[id*3] - U[id*3+1]*U[id*3+1]);
	m_eq[8] = rho[id] * U[id*3] * U[id*3+1];


	
	// Distribucion en espacio de momentos. m = M*field[id]

	for( uint i = 0 ; i < mesh->Q ; i++ ) {

	    m[i] = 0;

	    for( uint j = 0 ; j < mesh->Q ; j++ ) {

		m[i] += mesh->lattice.M[i*mesh->Q + j] * field[ id*mesh->Q + j ];

	    }

	}


      

	
	// Collision in momentum space
	
	for( uint k = 0 ; k < mesh->Q ; k++ )
	    m[k] = m[k]  -  1*( m[k] - m_eq[k] );
	    

	
	
	// Vuelta al espacio de fases. field = invM * m

	for( uint i = 0 ; i < mesh->Q ; i++ ) {

	    field[id*mesh->Q + i] = 0;

	    for( uint j = 0 ; j < mesh->Q ; j++ ) {

		field[id*mesh->Q + i] += mesh->lattice.invM[i*mesh->Q + j] * m[j];

	    }

	}
	

	

    }

}
