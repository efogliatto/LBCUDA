#include "hip/hip_runtime.h"
#include <cudaExampleCollision.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>

#include <cudaThreadedMatMult.h>



extern "C" __global__ void cudaExampleCollision( cuscalar* field, cuscalar* rho, cuscalar* U, cuscalar* Tau, cuscalar* M, cuscalar* invM, uint np, uint Q ) {
    

    int id = threadIdx.x + blockIdx.x*blockDim.x;

   
    if( id < np ) {


	// Distribuciones parciales
    
	cuscalar m[9];   // m:  Distribucion en espacio de momentos
    
	cuscalar m_eq[9];   // meq: Distribucion de equilibrio en espacio de momentos




	// Magnitud de la velocidad
	
	cuscalar ux = U[id*3], uy = U[id*3 + 1], uz = U[id*3 + 2];

	cuscalar umag = ux*ux + uy*uy + uz*uz;
	


	
	// Distribucion de equilibrio en espacio de momentos

	cuscalar localRho = rho[id];
	
	m_eq[0] = localRho;
	m_eq[1] = localRho * (-2 + 3*umag);
	m_eq[2] = localRho * (1 - 3*umag);
	m_eq[3] = localRho * ux;
	m_eq[4] = localRho * (-ux);
	m_eq[5] = localRho * uy;
	m_eq[6] = localRho * (-uy);
	m_eq[7] = localRho * (ux*ux - uy*uy);
	m_eq[8] = localRho * ux * uy;


	
	// Distribucion en espacio de momentos. m = M*field[id]

	for( uint i = 0 ; i < Q ; i++ ) {

	    m[i] = 0;

	    for( uint j = 0 ; j < Q ; j++ ) {

		m[i] += M[i*Q + j] * field[ id*Q + j ];

	    }

	}

	
	/* cudaThreadedMatMult<<<1,9>>>(M, field, id, Q); */

	/* __syncthreads(); */
	

	
	// Collision in momentum space
	
	for( uint k = 0 ; k < Q ; k++ )
	    m[k] = m[k]  -  Tau[k]*( m[k] - m_eq[k] );
	    

	
	
	// Vuelta al espacio de fases. field = invM * m

	for( uint i = 0 ; i < Q ; i++ ) {

	    field[id*Q + i] = 0;

	    for( uint j = 0 ; j < Q ; j++ ) {

		field[id*Q + i] += invM[i*Q + j] * m[j];

	    }

	}
	


    }

}
