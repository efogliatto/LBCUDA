#include "hip/hip_runtime.h"
#include <cudaExampleVelocity.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaExampleVelocity(cuscalar* field, cuscalar* rho, cuscalar* U, int* lvel, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    printf("%d\n",idx);    
   
    if( idx < np ) {


	// Velocidad local
	
	cuscalar lv[3] = {0,0,0};



	// Indice sobre componentes de velocidad
	
	for( uint j = 0 ; j < 3 ; j++ ) {

	    
	    // Indice sobre componentes de la funcion de distribucion
	    
	    for( uint k = 0 ; k < Q ; k++ ) {

		lv[j] += lvel[3*k + j] * field[idx*Q + k];
		    
	    }
	    
	}


	// Divide por densidad
	
	for( uint j = 0 ; j < 3 ; j++ ) {

	    lv[j] = lv[j] / rho[idx];
	
	}


	
	// Copa al arreglo global
	
	for( uint j = 0 ; j < 3 ; j++ ) {
	
	    U[idx*Q + j] = lv[j];
	
	}


	
    }

}
