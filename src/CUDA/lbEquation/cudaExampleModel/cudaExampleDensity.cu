#include "hip/hip_runtime.h"
#include <cudaExampleDensity.h>

#include <hip/hip_runtime.h>

#include <stdio.h>

#include <stdlib.h>



extern "C" __global__ void cudaExampleDensity(cuscalar* field, cuscalar* rho, int np, int Q ) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   
    if( idx < np ) {

    	int j;

    	cuscalar sum = 0;

    	for( j = 0 ; j < Q ; j++ ) {

    	    sum += field[ idx*Q + j ];

    	}


    	rho[idx] = sum;
	
    }

}
