#include "hip/hip_runtime.h"
#include <cudaFuerzaPresionEOS.h>

#include <stdio.h>


extern "C" __global__ void cudaFuerzaPresionEOS(scalar* p, scalar* rho, scalar* T, int np, int a, int b) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;

    // Constante universal de los gases (preguntar en que unidades se debe colocar ya que este modelo es adimensional)

    scalar R = 1; 

    // Calculo de la presion
    
    if( idx < np ) {

	p[idx] = ( ( rho[idx] * R * T[idx] ) / ( 1 - rho[idx] * b ) ) - ( a * rho[idx] * rho[idx] ) ;	    

	
    }
    

}
