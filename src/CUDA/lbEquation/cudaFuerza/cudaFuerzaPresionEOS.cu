#include "hip/hip_runtime.h"
#include <cudaFuerzaPresionEOS.h>

#include <stdio.h>


extern "C" __global__ void cudaFuerzaPresionEOS(scalar* p, scalar* rho, scalar* T, basicMesh* mesh, int a, int b) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;

    // Constante universal de los gases (preguntar en que unidades se debe colocar ya que este modelo es adimensional)

    scalar R = 1; 

    // Calculo de la presion
    
    if( idx < mesh->nPoints ) {

	p[idx] = ( ( rho[idx] * R * T[idx] ) / ( 1 - rho[idx] * b ) ) - ( a * rho[idx] * rho[idx] ) ;	    

	
    }
    

}
