#include "hip/hip_runtime.h"
#include <cudaFuerzaPsi.h>

#include <cudaFuerzaPresionEOS.h>

#include <stdio.h>

#include <math.h>


__global__ void cudaFuerzaPsi( cuscalar psi, cuscalar p_EOS, scalar rho, scalar c, scalar cs_2, scalar G) {

    // Suma de todas las componentes
    
	psi = pow ( fabs (( 2 * ( p_EOS - rho * cs_2 ) / ( G * c * c ) ) ), 0.5 );


}
