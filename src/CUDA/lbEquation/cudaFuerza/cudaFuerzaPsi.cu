#include "hip/hip_runtime.h"
#include <cudaFuerzaPsi.h>

#include <cudaFuerzaPresionEOS.h>

#include <stdio.h>

#include <math.h>


extern "C" __global__ void cudaFuerzaPsi(scalar* psi, scalar* p_EOS, scalar* rho, scalar c, scalar cs_2, scalar G, basicMesh* mesh) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;

    // Suma de todas las componentes
    
    if( idx <  mesh->nPoints ) {

	psi[idx] = pow ( ( 2 * ( p_EOS[idx] - rho[idx] * cs_2 ) / ( G * c * c ) ), 0.5 );

   }
    

}
