#include "hip/hip_runtime.h"
#include <cudaFuerzaPsi.h>

#include <cudaFuerzaPresionEOS.h>

#include <stdio.h>

#include <math.h>


extern "C" __device__ void cudaFuerzaPsi(cuscalar* psi, cuscalar p_EOS, cuscalar rho, cuscalar c, cuscalar cs_2, cuscalar G) {

    // Suma de todas las componentes
    
   *psi = pow ( ( 2 * ( p_EOS - rho * cs_2 ) / ( G * c * c ) ), 0.5 );

}
