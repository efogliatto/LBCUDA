#include "hip/hip_runtime.h"
#include <cudaFuerzaPsi.h>

#include <cudaFuerzaPresionEOS.h>

#include <stdio.h>

#include <math.h>


__device__ void cudaFuerzaPsi( cuscalar* psi, cuscalar p_EOS, cuscalar rho, cuscalar c, cuscalar cs_2, cuscalar G) {

    // Suma de todas las componentes
    
	*psi = pow ( fabs (( 2 * ( p_EOS - rho * cs_2 ) / ( G * c * c ) ) ), 0.5 );


}
