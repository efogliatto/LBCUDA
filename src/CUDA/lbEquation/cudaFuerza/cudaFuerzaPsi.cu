#include "hip/hip_runtime.h"
#include <cudaFuerzaPsi.h>

#include <cudaFuerzaPresionEOS.h>

#include <stdio.h>

#include <math.h>


extern "C" __global__ void cudaFuerzaPsi(scalar* psi, scalar* p_EOS, scalar* rho, scalar c, scalar cs_2, scalar G, int np) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;

    // Suma de todas las componentes
    
    if( idx <  np ) {

	psi[idx] = pow ( ( 2 * ( p_EOS[idx] - rho[idx] * cs_2 ) / ( G * c * c ) ), 0.5 );

   }
    

}
