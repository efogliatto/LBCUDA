#include "hip/hip_runtime.h"
#include <cudaFuerzaFuerzatotal.h>

#include <stdio.h>

extern "C" __global__ void cudaFuerzaFuerzatotal(cuscalar* f, cuscalar* fint, cuscalar* rho, cuscalar* g, int np) {


    int idx = threadIdx.x + blockIdx.x*blockDim.x;
	
    // Suma de todas las componentes

    
    if( idx < np ) {

	// Volumetric force add Interaction force

	uint j = 0 ;

	while( j < 3 ) {
	
	    f[idx*3+j] = rho[idx] * g[j] + fint[idx*3+j];

	    j++;
			
	}

    }

}
