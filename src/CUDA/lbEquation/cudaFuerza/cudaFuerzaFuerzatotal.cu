#include "hip/hip_runtime.h"
#include <cudaFuerzaFuerzatotal.h>

#include <stdio.h>

extern "C" __global__ void cudaFuerzaFuerzatotal(cuscalar* f, cuscalar* fint, cuscalar* rho, cuscalar* g, int np) {

	printf("ENTRE A LA FUNCION FUERZA TOTAL \n \n");


    int idx = threadIdx.x + blockIdx.x*blockDim.x;
	
    // Suma de todas las componentes

    if( idx < np ) {

	// Volumetric force add Interaction force

		uint j = 0 ;

		while( j < 3 ) {
	
	    	f[idx*3+j] = rho[idx] * g[j] + fint[idx*3+j];

	    	j++;
			
			printf("%lf \t",f[idx*3+j]);
		}
		printf("\n");
    }

}
