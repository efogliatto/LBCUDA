#include "hip/hip_runtime.h"
#include <cudaFuerzaFuerzatotal.h>

#include <stdio.h>









extern "C" __global__ void cudaFuerzaFuerzatotal(scalar* f, scalar* fint, scalar* rho, scalar* g, basicMesh* mesh) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
	
    // Suma de todas las componentes

    if( idx < mesh->nPoints ) {


	// Volumetric force add Interaccion force

	uint j = 0 ;

	while( j < 3 ) {
	
	    f[idx*3+j] = rho[idx] * g[j] + fint[idx*3+j];

	    j++;
	
	}

    }

}
