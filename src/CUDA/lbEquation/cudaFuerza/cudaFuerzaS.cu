#include "hip/hip_runtime.h"
#include <cudaFuerzaS.h>

#include <stdio.h>

#include <math.h>

extern "C" __global__ void cudaFuerzaS(scalar* S, scalar* f, scalar* fint, scalar* U, scalar* psi, scalar sigma, scalar tau_e, scalar tau_chi, basicMesh* mesh, scalar delta_t) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;

	// Valores de los pesos del modelo D2Q9


    if( idx < mesh->nPoints ) {



	scalar fintmag = 0.0;
	scalar dot = 0.0;

	uint k = 0 ;

	while( k < 3 )	{

	    fintmag += fint[idx*3 + k] * fint[idx*3 + k];

	    dot += U[idx*3 + k] * f[idx*3 + k];
	
	    k++;

	}

	scalar aux = 12 * sigma * fabsf( fintmag ) / ( psi[idx] * psi[idx] * delta_t );

	S[idx * mesh->Q + 0] = 0.0 ;	
	S[idx * mesh->Q + 1] = 6 * dot + ( aux / (tau_e - 0.5) )  ;    
	S[idx * mesh->Q + 2] = - 6 * dot - ( aux / (tau_chi - 0.5) )  ;
	S[idx * mesh->Q + 3] = f[0];
	S[idx * mesh->Q + 4] = - f[0];
	S[idx * mesh->Q + 5] = f[1];
	S[idx * mesh->Q + 6] = - f[1];
	S[idx * mesh->Q + 7] = 2 * ( U[0] * f[0] + U[1] * f[1] );
	S[idx * mesh->Q + 8] = ( U[0] * f[1] + U[1] * f[0] );

    }

}
