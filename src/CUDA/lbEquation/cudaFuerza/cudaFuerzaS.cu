#include "hip/hip_runtime.h"
#include <cudaFuerzaS.h>

#include <stdio.h>

#include <math.h>

__global__ void cudaFuerzaS(cuscalar* S, cuscalar* f, cuscalar* fint, cuscalar* U, cuscalar psi, cuscalar sigma, cuscalar* tau, scalar delta_t) {

	cuscalar fintmag = 0.0;
	cuscalar dot = 0.0;

	uint k = 0 ;

	while( k < 3 )	{

	    fintmag += fint[k] * fint[k];

	    dot += U[k] * f[k];
	
	    k++;

	}

	cuscalar aux = 12 * sigma * fabsf( fintmag ) / ( psi * psi * delta_t );

	S[0] = 0.0 ;	
	S[1] = 6 * dot + ( aux / (tau[1] - 0.5) )  ;    
	S[2] = - 6 * dot - ( aux / (tau[2] - 0.5) )  ;
	S[3] = f[0];
	S[4] = - f[0];
	S[5] = f[1];
	S[6] = - f[1];
	S[7] = 2 * ( U[0] * f[0] + U[1] * f[1] );
	S[8] = ( U[0] * f[1] + U[1] * f[0] );

}







