#include "hip/hip_runtime.h"
#include <cudaFuerzaS.h>

#include <stdio.h>

#include <math.h>

extern "C" __global__ void cudaFuerzaS(scalar* S, scalar* f, scalar* fint, scalar* U, scalar* psi, scalar sigma, scalar tau_e, scalar tau_chi, scalar delta_t, uint np, uint Q) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;

	// Valores de los pesos del modelo D2Q9


    if( idx < np ) {



	scalar fintmag = 0.0;
	scalar dot = 0.0;

	uint k = 0 ;

	while( k < 3 )	{

	    fintmag += fint[idx*3 + k] * fint[idx*3 + k];

	    dot += U[idx*3 + k] * f[idx*3 + k];
	
	    k++;

	}

	scalar aux = 12 * sigma * fabsf( fintmag ) / ( psi[idx] * psi[idx] * delta_t );

	S[idx * Q + 0] = 0.0 ;	
	S[idx * Q + 1] = 6 * dot + ( aux / (tau_e - 0.5) )  ;    
	S[idx * Q + 2] = - 6 * dot - ( aux / (tau_chi - 0.5) )  ;
	S[idx * Q + 3] = f[0];
	S[idx * Q + 4] = - f[0];
	S[idx * Q + 5] = f[1];
	S[idx * Q + 6] = - f[1];
	S[idx * Q + 7] = 2 * ( U[0] * f[0] + U[1] * f[1] );
	S[idx * Q + 8] = ( U[0] * f[1] + U[1] * f[0] );

    }

}
