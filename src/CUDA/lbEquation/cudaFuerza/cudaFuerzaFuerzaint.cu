#include "hip/hip_runtime.h"
#include <cudaFuerzaFuerzaint.h>

#include <cudaFuerzaPsi.h>

#include <stdio.h>


extern "C" __global__ void cudaFuerzaFuerzaint(scalar* fint, scalar* psi, int np, int Q,  int* lvel,int* nb, scalar G) {


       int idx = threadIdx.x + blockIdx.x*blockDim.x;

	// Valores de los pesos del modelo D2Q9

	scalar weight[9];

	weight[0] = 0.0 ;	
	weight[1] = (1/3);    
	weight[2] = (1/3);
	weight[3] = (1/3);
	weight[4] = (1/3);
	weight[5] = (1/12);
	weight[6] = (1/12);
	weight[7] = (1/12);
	weight[8] = (1/12);

	
    // Suma de todas las componentes
    
    if(  idx < np ) {


	// Local force
	
	scalar lf[3] = {0,0,0};


	// Move over velocity components
	
	uint j = 0 ;

	while( j < 3 ) {

	    // Move over model velocities

	    uint k = 0 ;

	    while( k < Q ) {

		lf[j] += (scalar)lvel[k*3+j] * weight[k] * psi[ nb[idx * Q + k] ];
		
		k++;
    
	    }
	
	    lf[j] = ( - G ) * lf[j]  * psi[idx];     
	    
            j++;

	}


	// Copy to global array
	
	j =0;

	while( j < 3 ) {
	
	    fint[idx*3+j] = lf[j];

	    j++;
	
	}

    }

}
