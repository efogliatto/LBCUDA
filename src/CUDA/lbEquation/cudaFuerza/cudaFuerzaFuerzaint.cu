#include "hip/hip_runtime.h"
#include <cudaFuerzaFuerzaint.h>

#include <cudaFuerzaPsi.h>

#include <stdio.h>


extern "C" __global__ void cudaFuerzaFuerzaint(scalar* fint, scalar* psi, basicMesh* mesh, scalar G) {


       int idx = threadIdx.x + blockIdx.x*blockDim.x;

	// Valores de los pesos del modelo D2Q9

	scalar weight[9];

	weight[0] = 0.0 ;	
	weight[1] = (1/3);    
	weight[2] = (1/3);
	weight[3] = (1/3);
	weight[4] = (1/3);
	weight[5] = (1/12);
	weight[6] = (1/12);
	weight[7] = (1/12);
	weight[8] = (1/12);

	
    // Suma de todas las componentes
    
    if(  idx < mesh->nPoints ) {


	// Local force
	
	scalar lf[3] = {0,0,0};


	// Move over velocity components
	
	uint j = 0 ;

	while( j < 3 ) {

	    // Move over model velocities

	    uint k = 0 ;

	    while( k < mesh->Q ) {

		lf[j] += (scalar)mesh->lattice.vel[k*3+j] * weight[k] * psi[ mesh->nb[idx][k] ];
		
		k++;
    
	    }
	
	    lf[j] = ( - G ) * lf[j]  * psi[idx];     
	    
            j++;

	}


	// Copy to global array
	
	j =0;

	while( j < 3 ) {
	
	    fint[idx*3+j] = lf[j];

	    j++;
	
	}

    }

}
