#include "hip/hip_runtime.h"
#include <cudaFuerzaFuerzaint.h>

#include <cudaFuerzaPresionEOS.h>

#include <cudaFuerzaPsi.h>

#include <stdio.h>


extern "C" __global__ void cudaFuerzaFuerzaint(cuscalar* fint, cuscalar* rho, cuscalar* T, int np, int Q,  int* lvel,int* nb, cuscalar G, cuscalar c, cuscalar cs_2, int a, int b) {


    int idx = threadIdx.x + blockIdx.x*blockDim.x;

	// Valores de los pesos del modelo D2Q9

	//cuscalar weight[(const int) Q];
	cuscalar weight[9];

	weight[0] = 0.0 ;	
	weight[1] = (1.0/3.0);    
	weight[2] = (1.0/3.0);
	weight[3] = (1.0/3.0);
	weight[4] = (1.0/3.0);
	weight[5] = (1.0/12.0);
	weight[6] = (1.0/12.0);
	weight[7] = (1.0/12.0);
	weight[8] = (1.0/12.0);

	
    // Suma de todas las componentes
    
    if( idx < np ) {

		if ( idx == 0 ) {
	
			printf("\n Fint calculada con CUDA \n\n");

			printf(" i: %d\n",idx);
		}
		// Local force
		
		cuscalar lf[3] = {0,0,0};

		// Move over velocity components
		uint j = 0;

		while( j < 3 ) {
			if ( idx == 0 ) 						
				printf("\t j: %d\n",j);

			cuscalar p_EOS = 0.0;
			
			cuscalar psi = 0.0;

			// Move over model velocities
			
			uint k = 0;

			//printf("\t Q: %d\n",Q);

			while( k < Q ) {

				int idx_nb = nb[idx * Q + k];	// index of neighbour to analize

				if ( idx_nb >= 0){  	//se podra cambiar por un while ? en caso de que si, ver como cambiar el else 
					
					cudaFuerzaPresionEOS( &p_EOS, rho[idx_nb] , T[idx_nb], a, b); 

					cudaFuerzaPsi( &psi, p_EOS, rho[idx], c, cs_2, G);
					
					lf[j] += (cuscalar)lvel[k*3+j] * weight[k] * psi ;	
				}

				else {
					lf[j] += 0.0;
				}
				if ( idx == 0 ) 
					printf("\t\t K:%d \t idx_nb:%d \t rho:%f \t p_EOS:%f \t psi:%f \t lvel:%f \t weight:%f \t lf:%f   \n",k,idx_nb,rho[idx],p_EOS,psi,(cuscalar)lvel[k*3+j],weight[k],lf[j]);

				k++;
			}

			// Se necesita utilizar el psi del nodo --> i que es en el que me encuentro

			cudaFuerzaPresionEOS( &p_EOS, rho[idx] , T[idx], a, b); 

			cudaFuerzaPsi( &psi, p_EOS, rho[idx], c, cs_2, G);
		
			lf[j] = ( - G ) * lf[j]  * psi;     
			
			j++;

		}


		// Copy to global array
		
		j = 0;
		
		while( j < 3 ) {
			fint[idx*3+j] = lf[j];
		
		//	printf("%lf \t",fint[i*3+j]);
		
			j++;
	
		}

		//printf("\n");

	}
	

	//printf("\nESTOY AFUERA DE FINT ------------\n\n");

}








