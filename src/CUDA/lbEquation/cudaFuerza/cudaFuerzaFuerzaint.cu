#include "hip/hip_runtime.h"
#include <cudaFuerzaFuerzaint.h>

#include <cudaFuerzaPresionEOS.h>

#include <cudaFuerzaPsi.h>

#include <stdio.h>


extern "C" __global__ void cudaFuerzaFuerzaint(cuscalar* fint, cuscalar* rho, cuscalar* T, int np, int Q,  int* lvel,int* nb, cuscalar G, cuscalar c, cuscalar cs_2, int a, int b) {


    int idx = threadIdx.x + blockIdx.x*blockDim.x;

	// Valores de los pesos del modelo D2Q9

	//const int aux = (const int)Q;
	const int aux = 9;

	scalar weight[aux];

	weight[0] = 0.0 ;	
	weight[1] = (1.0/3.0);    
	weight[2] = (1.0/3.0);
	weight[3] = (1.0/3.0);
	weight[4] = (1.0/3.0);
	weight[5] = (1.0/12.0);
	weight[6] = (1.0/12.0);
	weight[7] = (1.0/12.0);
	weight[8] = (1.0/12.0);

	
    // Suma de todas las componentes
    
    if(  idx < np ) {

		// Local force
	
		scalar lf[3] = {0,0,0};


		// Move over velocity components
	
		uint j = 0 ;

		while( j < 3 ) {

	    // Move over model velocities

	    	uint k = 0 ;

			cuscalar p_EOS = 0.0;
			cuscalar psi = 0.0;

	    	while( k < Q ) {

				int idx_nb = nb[idx * Q + k];	// index of neighbour to analize

				if ( idx_nb >= 0){
					
					cudaFuerzaPresionEOS( &p_EOS, rho[idx_nb] , T[idx_nb], a, b); 

					cudaFuerzaPsi( &psi, p_EOS, rho[idx], c, cs_2, G);
	
					
					lf[j] += (cuscalar)lvel[k*3+j] * weight[k] * psi ;
				}

				else {
					lf[j] += 0.0;
				}
		
				k++;
    		}	
			
			cudaFuerzaPresionEOS( &p_EOS, rho[idx] , T[idx], a, b); 

			cudaFuerzaPsi( &psi, p_EOS, rho[idx], c, cs_2, G);
	
		    lf[j] = ( - G ) * lf[j]  * psi;     
	    
            j++;

		}


	// Copy to global array
	
		j =0;

		while( j < 3 ) {
	
	    	fint[idx*3+j] = lf[j];

	    	j++;
	
		}

    }
}
