#include "hip/hip_runtime.h"
#include <cudaFuerzaFuerzaint.h>

#include <cudaFuerzaPresionEOS.h>

#include <cudaFuerzaPsi.h>

#include <stdio.h>


extern "C" __global__ void cudaFuerzaFuerzaint(cuscalar* fint, cuscalar* rho, cuscalar* T, int np, int Q,
					       int* lvel, int* reverse, int* nb, cuscalar G, cuscalar c, cuscalar cs_2, cuscalar a, cuscalar b) {


    int id = threadIdx.x + blockIdx.x*blockDim.x;

    
    if( id < np ) {

	
	// Valores de los pesos del modelo D2Q9

	scalar weight[9];

	weight[0] = 0.0 ;	
	weight[1] = (1.0/3.0);    
	weight[2] = (1.0/3.0);
	weight[3] = (1.0/3.0);
	weight[4] = (1.0/3.0);
	weight[5] = (1.0/12.0);
	weight[6] = (1.0/12.0);
	weight[7] = (1.0/12.0);
	weight[8] = (1.0/12.0);



	// Ver si el nodo tiene todos sus vecinos

	uint noNeigh = 0;

	for( uint k = 1 ; k < Q ; k++ ) {

	    if( nb[id*Q+k] == -1 )
		noNeigh = 1;

	}



	// Solo calcular fuerza en nodos que no estan sobre la pared

	cuscalar lf[3] = {0,0,0};
	
	if( noNeigh == 0 ) {

		uint k = 1;

	    while( k < Q ) {


	    	// Nodo vecino
       
	    	int neighId = nb[ id*Q + reverse[k] ];

	    	cuscalar _rho = rho[neighId];

	    	cuscalar _T = T[neighId];


	    	// Presion de estado en nodo vecino

	    	cuscalar p_EOS = 0;

	    	cudaFuerzaPresionEOS( &p_EOS, _rho , _T, a, b);

	    	cuscalar psi = 0;

	    	cudaFuerzaPsi( &psi, p_EOS, _rho, c, cs_2, G);

		

	    	// Suma para cada componente
	    
	    	cuscalar alpha = weight[k] * psi;

	    
	    	for( uint j = 0 ; j < 3 ; j++ )
	    	    lf[j] +=  alpha * (cuscalar)lvel[k*3+j] ;

			k++;
	    }



	    // Constante extra. Potencial en nodo

	    cuscalar p_EOS = 0;
	    
	    cudaFuerzaPresionEOS( &p_EOS, rho[id] , T[id], a, b);

	    cuscalar psi = 0;

	    cudaFuerzaPsi( &psi, p_EOS, rho[id], c, cs_2, G);
		


	    // Suma directamente en arreglo
		
		uint j = 0;
		
	    while( j < 3 )	{
			fint[id*3+j] =  lf[j] * (-G) * psi;
			j++;
		}
	    

	}

	
	// Sin fuerza sobre nodo de frontera
	
	else {

		uint j = 3;
	    while( j < 3 )	{
			fint[id*3+j] =  0;
			j++;
		}
	}	


    }
    
    



}










/* extern "C" __global__ void cudaFuerzaFuerzaint(cuscalar* fint, cuscalar* rho, cuscalar* T, int np, int Q, */
/* 					       int* lvel, int* nb, cuscalar G, cuscalar c, cuscalar cs_2, cuscalar a, cuscalar b) { */


/*     int idx = threadIdx.x + blockIdx.x*blockDim.x; */
    

/*     // Valores de los pesos del modelo D2Q9 */

/*     //cuscalar weight[(const int) Q]; */
/*     cuscalar weight[9]; */

/*     weight[0] = 0.0 ;	 */
/*     weight[1] = (1.0/3.0);     */
/*     weight[2] = (1.0/3.0); */
/*     weight[3] = (1.0/3.0); */
/*     weight[4] = (1.0/3.0); */
/*     weight[5] = (1.0/12.0); */
/*     weight[6] = (1.0/12.0); */
/*     weight[7] = (1.0/12.0); */
/*     weight[8] = (1.0/12.0); */

/*     // Vemos si la matriz de vecinos esta bien */

/*     if (idx == 5){ */
/* 	printf("\n\nvecinos en CUDA\n\n"); */
/* 	for(uint r = 0 ; r < np ; r++) { */
/* 	    for(uint y = 0 ; y < Q ; y++) { */
/* 		printf("\t %d",nb[r*Q+y]); */
/* 	    } */
/* 	    printf("\n"); */
/* 	} */
/*     } */
	
/*     // Suma de todas las componentes */
    
/*     if( idx < np ) { */

/* 	if ( idx == 0 ) { */
	
/* 	    printf("\n Fint calculada con CUDA \n\n"); */

/* 	    printf(" i: %d\n",idx); */
/* 	} */
/* 	// Local force */
		
/* 	cuscalar lf[3] = {0,0,0}; */

/* 	// Move over velocity components */
/* 	uint j = 0; */

/* 	while( j < 3 ) { */
/* 	    if ( idx == 0 ) 						 */
/* 		printf("\t j: %d\n",j); */

/* 	    cuscalar p_EOS = 0.0; */
			
/* 	    cuscalar psi = 0.0; */

/* 	    // Move over model velocities */
			
/* 	    uint k = 0; */

/* 	    //printf("\t Q: %d\n",Q); */

/* 	    while( k < Q ) { */

/* 		int idx_nb = nb[idx * Q + k];	// index of neighbour to analize */

/* 		if ( idx_nb >= 0){  	//se podra cambiar por un while ? en caso de que si, ver como cambiar el else  */
					
/* 		    cudaFuerzaPresionEOS( &p_EOS, rho[idx_nb] , T[idx_nb], a, b);  */

/* 		    cudaFuerzaPsi( &psi, p_EOS, rho[idx], c, cs_2, G); */
					
/* 		    lf[j] += (cuscalar)lvel[k*3+j] * weight[k] * psi ;	 */
/* 		} */

/* 		else { */
/* 		    lf[j] += 0.0; */
/* 		} */
/* 		if ( idx == 0 && j == 0 )  */
/* 		    printf("\t\t K:%d \t idx_nb:%d \t rho:%f \t p_EOS:%f \t psi:%f \t lvel:%f \t weight:%f \t lf:%f   \n",k,idx_nb,rho[idx],p_EOS,psi,(cuscalar)lvel[k*3+j],weight[k],lf[j]); */

/* 		k++; */
/* 	    } */

/* 	    // Se necesita utilizar el psi del nodo --> i que es en el que me encuentro */

/* 	    cudaFuerzaPresionEOS( &p_EOS, rho[idx] , T[idx], a, b);  */

/* 	    cudaFuerzaPsi( &psi, p_EOS, rho[idx], c, cs_2, G); */
		
/* 	    lf[j] = ( - G ) * lf[j]  * psi;      */
			
/* 	    j++; */

/* 	} */


/* 	// Copy to global array */
		
/* 	j = 0; */
		
/* 	while( j < 3 ) { */
/* 	    fint[idx*3+j] = lf[j]; */
		
/* 	    //	printf("%lf \t",fint[i*3+j]); */
		
/* 	    j++; */
	
/* 	} */

/* 	//printf("\n"); */

/*     } */
	



/* } */
