#include <hostToDeviceMesh.h>


__host__ void hostToDeviceMesh(cudaBasicMesh* cmesh, basicMesh* hmesh) {


    /**************************************/
    /*              Points                */
    /**************************************/
    

    // First create one-dimensional array

    int* hostPoints = (int*)malloc( hmesh->nPoints * 3 * sizeof(int) );

    for( uint i = 0 ; i < hmesh->nPoints ; i++ ) {

	for( uint j = 0 ; j < 3 ; j++ ) {

	    hostPoints[ 3*i + j ] = hmesh->points[i][j];

	}

    }


    // Copy to device
    
    hipMalloc( (void**)&cmesh->points, hmesh->nPoints*3*sizeof(int) );

    hipMemcpy( cmesh->points, hostPoints, hmesh->nPoints*3*sizeof(int), hipMemcpyHostToDevice );

    free(hostPoints);






    /**************************************/
    /*            Neighbours              */
    /**************************************/


    // First create one-dimensional array

    int* hostNb = (int*)malloc( hmesh->nPoints * hmesh->Q * sizeof(int) );

    for( uint i = 0 ; i < hmesh->nPoints ; i++ ) {

	for( uint j = 0 ; j < hmesh->Q ; j++ ) {

	    hostNb[ hmesh->Q*i + j ] = hmesh->nb[i][j];

	}

    }


    // Copy to device
    
    hipMalloc( (void**)&cmesh->nb, hmesh->nPoints*hmesh->Q*sizeof(int) );

    hipMemcpy( cmesh->nb, hostNb, hmesh->nPoints*hmesh->Q*sizeof(int), hipMemcpyHostToDevice );

    free(hostNb);






    /**************************************/
    /*        Informacion de grilla       */
    /**************************************/    

    cmesh->lattice.model = hmesh->lattice.model;

    cmesh->lattice.d = hmesh->lattice.d;

    cmesh->lattice.q = hmesh->lattice.q;



    // Velocidades de grilla
    
    hipMalloc( (void**)&cmesh->lattice.vel, 3*hmesh->Q*sizeof(int) );

    hipMemcpy( cmesh->lattice.vel, hmesh->lattice.vel, 3*hmesh->Q*sizeof(int), hipMemcpyHostToDevice );
    

    
    // Indices inversos
    
    hipMalloc( (void**)&cmesh->lattice.reverse, hmesh->Q*sizeof(int) );

    hipMemcpy( cmesh->lattice.reverse, hmesh->lattice.reverse, hmesh->Q*sizeof(int), hipMemcpyHostToDevice );


    
    // Matriz de transformacion MRT
    
    hipMalloc( (void**)&cmesh->lattice.M, hmesh->Q*hmesh->Q*sizeof(int) );

    hipMemcpy( cmesh->lattice.M, hmesh->lattice.M, hmesh->Q*hmesh->Q*sizeof(int), hipMemcpyHostToDevice );


    
    // Inversa de matriz de transformacion MRT
    
    hipMalloc( (void**)&cmesh->lattice.invM, hmesh->Q*hmesh->Q*sizeof(int) );

    hipMemcpy( cmesh->lattice.invM, hmesh->lattice.invM, hmesh->Q*hmesh->Q*sizeof(int), hipMemcpyHostToDevice );        
    
    


    /**************************************/
    /*               Other                */
    /**************************************/    

    cmesh->nPoints = hmesh->nPoints;
    
    cmesh->Q = hmesh->Q;

    cmesh->D = hmesh->D;
    
}
