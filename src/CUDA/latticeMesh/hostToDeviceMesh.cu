#include <hostToDeviceMesh.h>


__host__ void hostToDeviceMesh(cudaBasicMesh* cmesh, basicMesh* hmesh) {


    /**************************************/
    /*              Points                */
    /**************************************/
    

    // First create one-dimensional array

    int* hostPoints = (int*)malloc( hmesh->nPoints * 3 * sizeof(int) );

    for( uint i = 0 ; i < hmesh->nPoints ; i++ ) {

	for( uint j = 0 ; j < 3 ; j++ ) {

	    hostPoints[ 3*i + j ] = hmesh->points[i][j];

	}

    }


    // Copy to device
    
    hipMalloc( (void**)&cmesh->points, hmesh->nPoints*3*sizeof(int) );

    hipMemcpy( cmesh->points, hostPoints, hmesh->nPoints*3*sizeof(int), hipMemcpyHostToDevice );

    free(hostPoints);






    /**************************************/
    /*            Neighbours              */
    /**************************************/


    // First create one-dimensional array

    int* hostNb = (int*)malloc( hmesh->nPoints * hmesh->Q * sizeof(int) );

    for( uint i = 0 ; i < hmesh->nPoints ; i++ ) {

	for( uint j = 0 ; j < hmesh->Q ; j++ ) {

	    hostNb[ hmesh->Q*i + j ] = hmesh->nb[i][j];

	}

    }


    // Copy to device
    
    hipMalloc( (void**)&cmesh->nb, hmesh->nPoints*hmesh->Q*sizeof(int) );

    hipMemcpy( cmesh->nb, hostNb, hmesh->nPoints*hmesh->Q*sizeof(int), hipMemcpyHostToDevice );

    free(hostNb);






    /**************************************/
    /*        Informacion de grilla       */
    /**************************************/    

    cmesh->lattice.model = hmesh->lattice.model;

    cmesh->lattice.d = hmesh->lattice.d;

    cmesh->lattice.q = hmesh->lattice.q;



    // Velocidades de grilla
    
    hipMalloc( (void**)&cmesh->lattice.vel, 3*hmesh->Q*sizeof(int) );

    hipMemcpy( cmesh->lattice.vel, hmesh->lattice.vel, 3*hmesh->Q*sizeof(int), hipMemcpyHostToDevice );
    

    
    // Indices inversos
    
    hipMalloc( (void**)&cmesh->lattice.reverse, hmesh->Q*sizeof(int) );

    hipMemcpy( cmesh->lattice.reverse, hmesh->lattice.reverse, hmesh->Q*sizeof(int), hipMemcpyHostToDevice );


    
    // Matriz de transformacion MRT
    
    hipMalloc( (void**)&cmesh->lattice.M, hmesh->Q*hmesh->Q*sizeof(int) );

    hipMemcpy( cmesh->lattice.M, hmesh->lattice.M, hmesh->Q*hmesh->Q*sizeof(int), hipMemcpyHostToDevice );


    
    // Inversa de matriz de transformacion MRT
    
    hipMalloc( (void**)&cmesh->lattice.invM, hmesh->Q*hmesh->Q*sizeof(int) );

    hipMemcpy( cmesh->lattice.invM, hmesh->lattice.invM, hmesh->Q*hmesh->Q*sizeof(int), hipMemcpyHostToDevice );        
    
    


    /**************************************/
    /*               Other                */
    /**************************************/    

    cmesh->nPoints = hmesh->nPoints;
    
    cmesh->Q = hmesh->Q;

    cmesh->D = hmesh->D;




    /**************************************/
    /*              Fronteras             */
    /**************************************/


    // Cantidad de fronteras
    
    cmesh->bd.nbd = hmesh->bd.nbd;


    // Numero de elementos por frontera

    hipMalloc( (void**)&cmesh->bd.nbdelem, hmesh->bd.nbd*sizeof(uint) );

    hipMemcpy( cmesh->bd.nbdelem, hmesh->bd.nbdelem, hmesh->bd.nbd*sizeof(uint), hipMemcpyHostToDevice );


    
    // Indice de nodos para cada frontera

    {

	// Cantidad maxima de elementos por frontera
	
	uint count = 0;

	for(uint i = 0 ; i < hmesh->bd.nbd ; i++) {

	    if( hmesh->bd.nbdelem[i] > count )	    
		count = hmesh->bd.nbdelem[i];

	}

	cmesh->bd.maxCount = count;

	hmesh->bd.maxCount = count;


	// Primero arreglo unidimensional

	int* hostBdPoints = (int*)malloc( count * hmesh->bd.nbd * sizeof(int) );

	for( uint i = 0 ; i < hmesh->bd.nbd ; i++ ) {

	    for( uint j = 0 ; j < count ; j++ ) {

		if( j < hmesh->bd.nbdelem[i] ) {

		    hostBdPoints[i*count + j] = hmesh->bd.bdPoints[i][j];

		}

		else {

		    hostBdPoints[i*count + j] = -1;

		}
				

	    }

	}
	

	hipMalloc( (void**)&cmesh->bd.bdPoints, count * hmesh->bd.nbd * sizeof(int) );

	hipMemcpy( cmesh->bd.bdPoints, hostBdPoints, count * hmesh->bd.nbd * sizeof(int), hipMemcpyHostToDevice );
	
    }
    
    

    
}
