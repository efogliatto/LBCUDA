#include <hostToDeviceMesh.h>

__host__ void hostToDeviceMesh(cudaBasicMesh* cmesh, basicMesh* hmesh) {


    /**************************************/
    /*              Points                */
    /**************************************/
    

    // First create one-dimensional array

    int* hostPoints = (int*)malloc( hmesh->nPoints * 3 * sizeof(int) );

    for( uint i = 0 ; i < hmesh->nPoints ; i++ ) {

	for( uint j = 0 ; j < 3 ; j++ ) {

	    hostPoints[ 3*i + j ] = hmesh->points[i][j];

	}

    }


    // Copy to device
    
    hipMalloc( (void**)&cmesh->points, hmesh->nPoints*3*sizeof(int) );

    hipMemcpy( &cmesh->points, hostPoints, hmesh->nPoints*3*sizeof(int), hipMemcpyHostToDevice );

    free(hostPoints);






    /**************************************/
    /*            Neighbours              */
    /**************************************/


    // First create one-dimensional array

    int* hostNb = (int*)malloc( hmesh->nPoints * hmesh->Q * sizeof(int) );

    for( uint i = 0 ; i < hmesh->nPoints ; i++ ) {

	for( uint j = 0 ; j < hmesh->Q ; j++ ) {

	    hostNb[ 3*i + j ] = hmesh->nb[i][j];

	}

    }


    // Copy to device
    
    hipMalloc( (void**)&cmesh->nb, hmesh->nPoints*hmesh->Q*sizeof(int) );

    hipMemcpy( &cmesh->nb, hostNb, hmesh->nPoints*hmesh->Q*sizeof(int), hipMemcpyHostToDevice );

    free(hostNb);





    /**************************************/
    /*               Other                */
    /**************************************/    

    cmesh->Q = hmesh->Q;

    cmesh->D = hmesh->D;
    
}
